#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_cooperative_groups.h>

#include "../../global_functions.h"
#include "programs/scenarios.h"
#include <nvtx3/nvToolsExt.h>

void initialization(int width, int height, int iter, double dx, double dy, int gpus, int compare, int overlap, int test, dim3 blockDim, dim3 gridDim){
    /*
    Variables            | Type        | Description
    total                | int         | The total number of elements within the matrix
    overlap_calc         | int         | Used to find how many elements less the kernelCollMid has to compute when we have overlap
    threadSize           | int         | Finds the total amount of threads in use
    
    device_nr            | int*        | Allows the GPU to know its GPU index

    rows_total           | int         | Total number of rows to be computed on
    rows_per_device      | int         | Number of rows per device, rounded down
    rows_leftover        | int         | Number of rows leftover when rounded down

    rows_device          | int*        | Rows to allocate on the GPU
    rows_compute_device  | int*        | Rows the GPU will compute on
    rows_staring_index   | int*        | Index on the CPU matrix that the first element of the GPU matrix belongs

    threadInformation[0] | int         | Number of computations per thread on GPU 0, rounded down
    threadInformation[1] | int         | Number of computations left over when rounded down
    threadInformation[2] | int         | Number of computations per thread on GPU n-1, rounded down, is used as if there are an unequal amount of rows between device, 
                                         the first and last GPU will certainly be in each group
    threadInformation[3] | int         | Number of computations left over when rounded down
    threadInformation[4] | int         | Number of computations per thread for 1 row, rounded down
    threadInformation[5] | int         | Number of computations left over for 1 row when rounded down

    mat                  | double*     | The matrix allocated on the CPU
    mat_gpu              | double**    | One of the matrices allocated on the GPU
    mat_gpu_tmp          | double**    | The other matrix allocated on the GPU

    kernelCollEdge       | void***     | The inputfeatures to the jacobiEdge GPU kernel
    kernelCollMid        | void***     | The inputfeatures to the jacobiMid GPU kernel
    */


    int total = width*height;
    int overlap_calc = (width-2)*overlap;
    int threadSize = blockDim.x*blockDim.y*blockDim.z*gridDim.x*gridDim.y*gridDim.z;

    int *device_nr;
    cudaErrorHandle(hipHostMalloc(&device_nr, gpus*sizeof(int*)));
    for(int g = 0; g < gpus; g++){
        device_nr[g] = g;
    }

    // Find general information about rows for all the GPUs
    int rows_total = height-2; // Removes upper and lower layer
    int rows_per_device = rows_total/gpus;
    int rows_leftover = rows_total%gpus;

    // Finds more specific information about rows for each GPU
    int *rows_device, *rows_compute_device, *rows_starting_index;
    cudaErrorHandle(hipHostMalloc(&rows_device, gpus*sizeof(int*)));
    cudaErrorHandle(hipHostMalloc(&rows_starting_index, gpus*sizeof(int*)));
    cudaErrorHandle(hipHostMalloc(&rows_compute_device, gpus*sizeof(int*)));
    // Calculate the number of rows for each device
    for (int g = 0; g < gpus; g++) {
        int extra_row = (g < rows_leftover) ? 1 : 0;
        rows_device[g] = rows_per_device + extra_row + 2; // Need to add ghost layers
        rows_compute_device[g] = rows_per_device + extra_row - (2*overlap); // Will compute 2 rows(upper and lower) on their own if overlap is used
        rows_starting_index[g] = g * rows_per_device + min(g, rows_leftover);
    }

    // Estimates elements per thread for 3 scenarios,
    // Scenario 1(0, 1) and 2(2, 3) are depending on if number of rows are unequal for the GPUs, which will mean that the first and last gpu has different amount of elements
    // Scenario 3(4, 5) is only used for overlap, where we only want to know the number of elements compared to threads for 1 row
    int *threadInformation;
    cudaErrorHandle(hipHostMalloc(&threadInformation, 6*sizeof(int)));
    threadInformation[0] = ((rows_compute_device[0])     *(width-2))/threadSize;
    threadInformation[1] = ((rows_compute_device[0])     *(width-2))%threadSize;
    threadInformation[2] = ((rows_compute_device[gpus-1])*(width-2))/threadSize;
    threadInformation[3] = ((rows_compute_device[gpus-1])*(width-2))%threadSize;
    threadInformation[4] = (1                            *(width-2))/threadSize;
    threadInformation[5] = (1                            *(width-2))%threadSize;

    double *mat;
    double **mat_gpu, **mat_gpu_tmp;
    cudaErrorHandle(hipHostMalloc(&mat,          total*sizeof(double)));
    cudaErrorHandle(hipHostMalloc(&mat_gpu,      gpus*sizeof(double*)));
    cudaErrorHandle(hipHostMalloc(&mat_gpu_tmp,  gpus*sizeof(double*)));

    // Fills up the mat matrix with starting values
    fillValues(mat, dx, dy, width, height);

    // Allocates memory on devices based on number of rows for each device
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMalloc(&mat_gpu[g],     width*rows_device[g]*sizeof(double)));
        cudaErrorHandle(hipMalloc(&mat_gpu_tmp[g], width*rows_device[g]*sizeof(double)));
    }

    void ***kernelCollEdge;
    cudaErrorHandle(hipHostMalloc(&kernelCollEdge, gpus * sizeof(void**)));
    // Allocates the elements in the kernelCollEdge, used for hipLaunchCooperativeKernel as functon variables.
    for (int g = 0; g < gpus; g++) {
        void **kernelArgs = new void*[7];
        kernelArgs[0] = &mat_gpu[g];
        kernelArgs[1] = &mat_gpu_tmp[g];
        kernelArgs[2] = &width;
        kernelArgs[3] = &height;
        kernelArgs[4] = &rows_compute_device[g];
        kernelArgs[5] = &threadInformation[4];
        kernelArgs[6] = &threadInformation[5];

        kernelCollEdge[g] = kernelArgs;
    }

    void ***kernelCollMid;
    cudaErrorHandle(hipHostMalloc(&kernelCollMid, gpus * sizeof(void**)));
    // Allocates the elements in the kernelCollMid, used for hipLaunchCooperativeKernel as functon variables.
    for (int g = 0; g < gpus; g++) {
        void **kernelArgs = new void*[12];
        kernelArgs[0] = &mat_gpu[g];     
        kernelArgs[1] = &mat_gpu_tmp[g];
        kernelArgs[2] = &width;
        kernelArgs[3] = &height;
        kernelArgs[4] = &rows_leftover;
        kernelArgs[5] = &device_nr[g];
        kernelArgs[6] = &rows_compute_device[g];
        kernelArgs[7] = &threadInformation[0];
        kernelArgs[8] = &threadInformation[1];
        kernelArgs[9] = &threadInformation[2];
        kernelArgs[10] = &threadInformation[3];
        kernelArgs[11] = &overlap_calc;

        kernelCollMid[g] = kernelArgs;
    }

    // Copies elements from the matrix located on the CPU to mat_gpu[g] matrix located on each GPU g
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMemcpy(mat_gpu[g], mat+rows_starting_index[g]*width, rows_device[g]*width*sizeof(double), hipMemcpyHostToDevice));
    }

    cudaErrorHandle(hipDeviceSynchronize());

    nvtxRangePushA("Area of Interest");
    if(gpus < 2){
        printf("You are running on less than 2 gpus, to be able to communicate between gpus you are required to compute on more than 1 gpu.\n");
    }
    else{
        if(overlap == 1){
            if(test == 0){
                full_calculation_overlap(mat_gpu, mat_gpu_tmp, height, width, iter, gpus, rows_device, gridDim, blockDim, kernelCollEdge, kernelCollMid);
            }
            else if(test == 1){
                no_kernel_overlap(mat_gpu, mat_gpu_tmp, height, width, iter, gpus, rows_device, gridDim, blockDim, kernelCollEdge, kernelCollMid);
            }
            else if(test == 2){
                no_communication_overlap(mat_gpu, mat_gpu_tmp, height, width, iter, gpus, rows_device, gridDim, blockDim, kernelCollEdge, kernelCollMid);
            }
            else if(test == 3){
                only_calculation_overlap(mat_gpu, mat_gpu_tmp, height, width, iter, gpus, rows_device, gridDim, blockDim, kernelCollEdge, kernelCollMid);
            }
            else if(test == 4){
                only_communication_overlap(mat_gpu, mat_gpu_tmp, height, width, iter, gpus, rows_device, gridDim, blockDim, kernelCollEdge, kernelCollMid);
            }
        }
        else{
            if(test == 0){
                full_calculation_nooverlap(mat_gpu, mat_gpu_tmp, height, width, iter, gpus, rows_device, gridDim, blockDim, kernelCollMid);
            }
            else if(test == 1){
                no_kernel_nooverlap(mat_gpu, mat_gpu_tmp, height, width, iter, gpus, rows_device, gridDim, blockDim, kernelCollMid);
            }
            else if(test == 2){
                no_communication_nooverlap(mat_gpu, mat_gpu_tmp, height, width, iter, gpus, rows_device, gridDim, blockDim, kernelCollMid);
            }
            else if(test == 3){
                only_calculation_nooverlap(mat_gpu, mat_gpu_tmp, height, width, iter, gpus, rows_device, gridDim, blockDim, kernelCollMid);
            }
            else if(test == 4){
                only_communication_nooverlap(mat_gpu, mat_gpu_tmp, height, width, iter, gpus, rows_device, gridDim, blockDim, kernelCollMid);
            }
        }
    }




    nvtxRangePop();



    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }

    // Copies each element from mat_gpu[g] located on each GPU g, to the mat matrix located on the CPU.
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMemcpyAsync(mat + (rows_starting_index[g]+1)*width, mat_gpu[g] + width, (rows_compute_device[g]+2*overlap)*width*sizeof(double), hipMemcpyDeviceToHost));
    }
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }


    // Used to compare the matrix to the matrix which only the CPU created
    if(compare == 1){
        double* mat_compare = (double*)malloc(width * height * sizeof(double));
        FILE *fptr;
        char filename[30];
        sprintf(filename, "../CPU/CPUMatrix%i_%i.txt", width, height);

        printf("Comparing the matrixes\n");

        fptr = fopen(filename, "r");
        if (fptr == NULL) {
            printf("Error opening file.\n");
            exit(EXIT_FAILURE);
        }

        // Read matrix values from the file
        for (int i = 0; i < height; i++) {
            for (int j = 0; j < width; j++) {
                if (fscanf(fptr, "%lf", &mat_compare[j + i * width]) != 1) {
                    printf("Error reading from file.\n");
                    fclose(fptr);
                    free(mat_compare);
                    exit(EXIT_FAILURE);
                }
            }
        }

        fclose(fptr);


        // Comparing the elements
        for (int i = 1; i < height-1; i++) {
            for (int j = 1; j < width-1; j++) {
                if (fabs(mat[j + i * width] - mat_compare[j + i * width]) > 1e-16)  {
                    printf("Mismatch found at position (%d, %d) (%.16f, %.16f)\n", i, j, mat[j + i * width], mat_compare[j + i * width]);
                    free(mat_compare);
                    exit(EXIT_FAILURE);
                    cudaErrorHandle(hipDeviceSynchronize());
                }
            }
        }


        printf("All elements match!\n");


        // Free allocated memory
        free(mat_compare);
    }



    // Frees up memory as we are finished with the program
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipFree(mat_gpu[g]));
        cudaErrorHandle(hipFree(mat_gpu_tmp[g]));
    }
    cudaErrorHandle(hipHostFree(mat));
    cudaErrorHandle(hipHostFree(mat_gpu));
    cudaErrorHandle(hipHostFree(mat_gpu_tmp));
    cudaErrorHandle(hipHostFree(threadInformation));
    cudaErrorHandle(hipHostFree(device_nr));
    cudaErrorHandle(hipHostFree(rows_device));
    cudaErrorHandle(hipHostFree(rows_starting_index));
    cudaErrorHandle(hipHostFree(rows_compute_device));
}



int main(int argc, char *argv[]) {
    /*
    Functions   | Type           | Input
    start       | void           | int width, int height, int iter, double dx, double dy, 
                                    int gpus, int compare, int overlap, int test, 
                                    dim3 blockDim, dim3 gridDim
    ____________________________________________________________________________
    Variables   | Type  | Description
    width       | int   | The width of the matrix
    height      | int   | The height of the matrix
    iter        | int   | Amount of iterations
    gpus        | int   | Number of gpus in use
    compare     | int   | If one wants to compare the output with a previously CPU computed matrix
    overlap     | int   | If one want to overlap or not
    test        | int   | Select which type of test one wants to run

    dx          | float | Used to give value to the elements of the matrix
    dy          | float | Used to give value to the elements of the matrix

    blockDim    | dim3  | Size of the threadblock
    gridDim     | dim3  | Size of the blockgrid

    For all true/false integers, 0 = false, 1 = true
    */

    // Checks if the correct amount of inputs is used
    if (argc != 8) {
        printf("Wrong amount of inputs: %s <width> <height> <iter> <gpus> <compare> <overlap> <test>", argv[0]);
        return 1;
    }

    int width = atoi(argv[1]);
    int height = atoi(argv[2]);
    int iter = atoi(argv[3]);
    int gpus = atoi(argv[4]);
    int compare = atoi(argv[5]);
    int overlap = atoi(argv[6]);
    int test = atoi(argv[7]);

    if(width < 1){
        printf("Width is to small\n");
    }
    else if(height < 1){
        printf("Heigth is to small\n");
    }
    else if(iter < 1){
        printf("To few selected iterations\n");
    }
    else if(gpus < 1){
        printf("Selected to few GPUs\n");
    }
    else if(compare > 1 || compare < 0){
        printf("Compare variable can only be\n"
                "0 - Do not compare the output matrix with previously created matrix\n"
                "1 - Compare with previously created matrix\n");
    }
    else if(overlap > 1 || overlap < 0){
        printf("You can only select the values\n"
                "0 - Do not overlap communication and computation\n"
                "1 - Overlap communication and computation\n");
    }
    else if(test > 4 || test < 0){
        printf("There is no test with the current value, please select one of the following.\n"
                "0 - Full Computation\n"
                "1 - No kernel\n"
                "2 - No Communication\n"
                "3 - Only computation\n"
                "4 - Only communication\n");
    }

    double dx = 2.0 / (width - 1);
    double dy = 2.0 / (height - 1);

    dim3 blockDim(32, 32, 1);
    dim3 gridDim(16, 1, 1);

    initialization(width, height, iter, dx, dy, gpus, compare, overlap, test, blockDim, gridDim);

    return 0;
}