#include "cuda_functions.h"

void initializeStreamsAndEvents(hipStream_t streams[2], hipEvent_t events[4], hipEvent_t *startevent, hipEvent_t *stopevent) {
    cudaErrorHandle(hipStreamCreate(&streams[0]));
    cudaErrorHandle(hipStreamCreate(&streams[1]));
    cudaErrorHandle(hipEventCreate(&events[0]));
    cudaErrorHandle(hipEventCreate(&events[1]));
    cudaErrorHandle(hipEventCreate(&events[2]));
    cudaErrorHandle(hipEventCreate(&events[3]));
    cudaErrorHandle(hipEventCreate(startevent));
    cudaErrorHandle(hipEventCreate(stopevent));
}

void freeStreamsAndEvents(hipStream_t streams[2], hipEvent_t events[4], hipEvent_t *startevent, hipEvent_t *stopevent) {
    cudaErrorHandle(hipStreamDestroy(streams[0]));
    cudaErrorHandle(hipStreamDestroy(streams[1]));
    cudaErrorHandle(hipEventDestroy(events[0]));
    cudaErrorHandle(hipEventDestroy(events[1]));
    cudaErrorHandle(hipEventDestroy(events[2]));
    cudaErrorHandle(hipEventDestroy(events[3]));

}
