#include <stdio.h>
#include <math.h>
#include <mpi.h>

#include "cuda_functions.h"
#include "jacobi.h"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

void full_calculation_overlap(double *data_gpu, double *data_gpu_tmp, int width, int height, int depth_node, int iter, int rank, int size, dim3 gridDim, dim3 blockDim, void** kernelMid, void** kernelEdge){
    hipStream_t streams[2];
    hipEvent_t events[2], startevent, stopevent;
    cudaErrorHandle(hipStreamCreate(&streams[0]));
    cudaErrorHandle(hipStreamCreate(&streams[1]));
    cudaErrorHandle(hipEventCreate(&events[0]));
    cudaErrorHandle(hipEventCreate(&events[1]));
    cudaErrorHandle(hipEventCreate(&startevent));
    cudaErrorHandle(hipEventCreate(&stopevent));
    MPI_Request myRequest[4];
    MPI_Status myStatus[4];
    double *data_cpu;
    cudaErrorHandle(hipHostMalloc(&data_cpu, 4*width*height*sizeof(double)));

    cudaErrorHandle(hipDeviceSynchronize());
    cudaErrorHandle(hipEventRecord(startevent));
    

    while(iter > 0){
        // Runs GPU Kernel
        cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiEdge, gridDim, blockDim, kernelEdge, 0, streams[0]));
        cudaErrorHandle(hipEventRecord(events[0], streams[0]));

        cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelMid, 0, streams[0]));
        cudaErrorHandle(hipEventRecord(events[1], streams[1]));


        cudaErrorHandle(hipStreamWaitEvent(streams[0], events[0]));
        cudaErrorHandle(hipStreamWaitEvent(streams[1], events[0]));

        if(rank == 0){
            cudaErrorHandle(hipMemcpy(data_cpu, data_gpu_tmp + (depth_node-2)*width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));
        }
        else if(rank == size-1){
            cudaErrorHandle(hipMemcpy(data_cpu + width*height, data_gpu_tmp + width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));
        }
        else{
            cudaErrorHandle(hipMemcpyAsync(data_cpu, data_gpu_tmp + (depth_node-2)*width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));
            cudaErrorHandle(hipEventRecord(events[0], streams[0]));
            cudaErrorHandle(hipMemcpy(data_cpu + width*height, data_gpu_tmp + width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));
            cudaErrorHandle(hipStreamWaitEvent(streams[0], events[0]));
        }

        if(rank == 0){
            MPI_Isend(&data_cpu[0],              width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[0]);
            MPI_Irecv(&data_cpu[3*width*height], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[1]); 
        }
        else if(rank == size-1){
            MPI_Irecv(&data_cpu[2*width*height], width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[0]); 
            MPI_Isend(&data_cpu[width*height],   width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[1]); 
        }
        else{
            MPI_Irecv(&data_cpu[2*width*height], width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[0]); 
            MPI_Isend(&data_cpu[width*height],   width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[1]); 

            MPI_Isend(&data_cpu[0],              width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[2]);
            MPI_Irecv(&data_cpu[3*width*height], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[3]); 
        }

        MPI_Waitall(rank == 0 || rank == size - 1 ? 2 : 4, myRequest, myStatus);

        if(rank == 0){
            cudaErrorHandle(hipMemcpy(data_gpu_tmp + (depth_node-1)*width*height, data_cpu + 3*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
        }
        else if(rank == size-1){
            cudaErrorHandle(hipMemcpy(data_gpu_tmp,                               data_cpu + 2*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
        }
        else{
            cudaErrorHandle(hipMemcpyAsync(data_gpu_tmp + (depth_node-1)*width*height, data_cpu + 3*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
            cudaErrorHandle(hipEventRecord(events[0], streams[0]));
            cudaErrorHandle(hipMemcpy(data_gpu_tmp,                               data_cpu + 2*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
            cudaErrorHandle(hipStreamWaitEvent(streams[0], events[0]));
        }


        cudaErrorHandle(hipStreamWaitEvent(streams[0], events[1]));
        cudaErrorHandle(hipStreamWaitEvent(streams[1], events[1]));


        double *data_change = data_gpu;
        data_gpu = data_gpu_tmp;
        data_gpu_tmp = data_change;
        
        void *temp_mid = kernelMid[0];
        kernelMid[0] = kernelMid[1];
        kernelMid[1] = temp_mid;

        void *temp_edge= kernelEdge[0];
        kernelEdge[0] = kernelEdge[1];
        kernelEdge[1] = temp_edge;

        iter--;
    }


    cudaErrorHandle(hipDeviceSynchronize());

    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    cudaErrorHandle(hipDeviceSynchronize());
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.5f s\n", milliseconds/1000);

    //freeStreamsAndEvents(1, streams, events, &startevent, &stopevent);
}

void full_calculation_nooverlap(double *data_gpu, double *data_gpu_tmp, int width, int height, int depth_node, int iter, int rank, int size, dim3 gridDim, dim3 blockDim, void** kernelMid){
    hipStream_t streams[2];
    hipEvent_t events, startevent, stopevent;
    cudaErrorHandle(hipStreamCreate(&streams[0]));
    cudaErrorHandle(hipStreamCreate(&streams[1]));
    cudaErrorHandle(hipEventCreate(&events));
    cudaErrorHandle(hipEventCreate(&startevent));
    cudaErrorHandle(hipEventCreate(&stopevent));
    MPI_Request myRequest[4];
    MPI_Status myStatus[4];
    double *data_cpu;
    cudaErrorHandle(hipHostMalloc(&data_cpu, 4*width*height*sizeof(double)));

    cudaErrorHandle(hipDeviceSynchronize());
    cudaErrorHandle(hipEventRecord(startevent));
    

    while(iter > 0){
        // Runs GPU Kernel
        cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelMid, 0, streams[0]));
        cudaErrorHandle(hipEventRecord(events, streams[0]));
        cudaErrorHandle(hipEventSynchronize(events));

        if(rank == 0){
            cudaErrorHandle(hipMemcpy(data_cpu, data_gpu_tmp + (depth_node-2)*width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));
        }
        else if(rank == size-1){
            cudaErrorHandle(hipMemcpy(data_cpu + width*height, data_gpu_tmp + width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));
        }
        else{
            cudaErrorHandle(hipMemcpy(data_cpu, data_gpu_tmp + (depth_node-2)*width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));
            cudaErrorHandle(hipMemcpy(data_cpu + width*height, data_gpu_tmp + width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));
        }

        if(rank == 0){
            MPI_Isend(&data_cpu[0],              width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[0]);
            MPI_Irecv(&data_cpu[3*width*height], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[1]); 
        }
        else if(rank == size-1){
            MPI_Irecv(&data_cpu[2*width*height], width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[0]); 
            MPI_Isend(&data_cpu[width*height],   width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[1]); 
        }
        else{
            MPI_Irecv(&data_cpu[2*width*height], width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[0]); 
            MPI_Isend(&data_cpu[width*height],   width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[1]); 

            MPI_Isend(&data_cpu[0],              width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[2]);
            MPI_Irecv(&data_cpu[3*width*height], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[3]); 
        }

        MPI_Waitall(rank == 0 || rank == size - 1 ? 2 : 4, myRequest, myStatus);

        if(rank == 0){
            cudaErrorHandle(hipMemcpy(data_gpu_tmp + (depth_node-1)*width*height, data_cpu + 3*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
        }
        else if(rank == size-1){
            cudaErrorHandle(hipMemcpy(data_gpu_tmp,                               data_cpu + 2*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
        }
        else{
            cudaErrorHandle(hipMemcpy(data_gpu_tmp + (depth_node-1)*width*height, data_cpu + 3*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
            cudaErrorHandle(hipMemcpy(data_gpu_tmp,                               data_cpu + 2*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
        }

        double *data_change = data_gpu;
        data_gpu = data_gpu_tmp;
        data_gpu_tmp = data_change;
        
        void *temp = kernelMid[0];
        kernelMid[0] = kernelMid[1];
        kernelMid[1] = temp;

        iter--;
    }


    cudaErrorHandle(hipDeviceSynchronize());

    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    cudaErrorHandle(hipDeviceSynchronize());
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.5f s\n", milliseconds/1000);

    //freeStreamsAndEvents(1, streams, events, &startevent, &stopevent);
}