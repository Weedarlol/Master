#include <stdio.h>
#include <math.h>
#include <mpi.h>

#include "cuda_functions.h"
#include "jacobi.h"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

void full_calculation(double *data_gpu, double *data_gpu_tmp, int width, int height, int depth_node, int iter, int slices_device, int rank, dim3 gridDim, dim3 blockDim, void** kernelCollMid){
    hipStream_t streams[2];
    hipEvent_t events, startevent, stopevent;
    cudaErrorHandle(hipStreamCreate(&streams[0]));
    cudaErrorHandle(hipStreamCreate(&streams[1]));
    cudaErrorHandle(hipEventCreate(&events));
    cudaErrorHandle(hipEventCreate(&startevent));
    cudaErrorHandle(hipEventCreate(&stopevent));
    MPI_Request myRequest[2];
    MPI_Status myStatus[2];
    double *data_cpu;
    cudaErrorHandle(hipHostMalloc(&data_cpu, 2*width*height*sizeof(double)));

    cudaErrorHandle(hipDeviceSynchronize());
    cudaErrorHandle(hipEventRecord(startevent));
    

    while(iter > 0){
        // Runs GPU Kernel
        cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobi, gridDim, blockDim, kernelCollMid, 0, streams[0]));
        cudaErrorHandle(hipEventRecord(events, streams[0]));
        cudaErrorHandle(hipEventSynchronize(events));

        // Copies data from GPU to CPU
        if(rank == 0){
            cudaErrorHandle(hipMemcpy(data_cpu, data_gpu_tmp + (depth_node-2)*width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));
        }
        else if(rank == 1){
            cudaErrorHandle(hipMemcpy(data_cpu, data_gpu_tmp + width*height,                width*height*sizeof(double), hipMemcpyDeviceToHost));
        }

        // Sends data from node 0 CPU to node 1 CPU
        if(rank == 0){
            MPI_Isend(&data_cpu[0],            width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[0]);
            MPI_Irecv(&data_cpu[width*height], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[1]); 
        }
        else if(rank == 1){
            MPI_Irecv(&data_cpu[width*height], width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[0]); 
            MPI_Isend(&data_cpu[0],            width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[1]); 
        }
        MPI_Waitall(2, myRequest, myStatus);

        // Copies data from CPU to GPU
        if(rank == 0){
            cudaErrorHandle(hipMemcpy(data_gpu_tmp + (depth_node-1)*width*height, data_cpu + width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
        }
        else if(rank == 1){
            cudaErrorHandle(hipMemcpy(data_gpu_tmp,                               data_cpu + width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
        }

        double *data_change = data_gpu;
        data_gpu = data_gpu_tmp;
        data_gpu_tmp = data_change;
        
        void *temp = kernelCollMid[0];
        kernelCollMid[0] = kernelCollMid[1];
        kernelCollMid[1] = temp;

        iter--;
    }

    printf("Finished full computation loop\n");

    cudaErrorHandle(hipDeviceSynchronize());

    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    cudaErrorHandle(hipDeviceSynchronize());
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.5f s\n", milliseconds/1000);

    //freeStreamsAndEvents(1, streams, events, &startevent, &stopevent);
}