#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>

#include "../../global_functions.h"

int main(int argc, char *argv[]) {
    /*
    width       | int    | The width of the grid
    height      | int    | The height of the 
    depth       | int    | The depth of the grid
    iter        | int    | Number of max iterations for the jacobian algorithm
    create_matrix| int    | Boolean for if one prints out the output matrix into a file or not. 1 = yes, 0 = no

    dx          | double | Distance between each element in the grid in x direction
    dy          | double | Distance between each element in the grid in y direction
    dz          | double | Distance between each element in the grid in z direction

    mat         |*double | Pointer to the grid
    mat_tmp     |*double | Pointer to the grid

    start       | clock_t| Starttime of time estimation
    end         | clock_t| Endtime of time estimation

    division    | double | Made a variable to not have to calculate it for each element
    */

    if (argc != 5) {
        printf("Usage: %s <Width> <Height> <Depth> <Iterations>", argv[0]);
        return 1;
    }

    int width = atoi(argv[1]);
    int height = atoi(argv[2]);
    int depth = atoi(argv[3]);
    int iter = atoi(argv[4]);
    int create_matrix = atoi(argv[5]);

    double dx = 2.0 / (width - 1);
    double dy = 2.0 / (height - 1);
    double dz = 2.0 / (depth - 1);

    double *mat;
    double *mat_tmp;

    clock_t start, end;

    mat = (double*)malloc(width*height*depth*sizeof(double));
    mat_tmp = (double*)malloc(width*height*depth*sizeof(double));

    // Fills up the mat grid with starting values
    fillValues3D(mat, width, height, depth, dx, dy, dz);

    double division = 1/6.0;
    start = clock();

    /* Performing Jacobian grid Calculation */
    // Performing a number of iterations while statement is not satisfied
    while (iter > 0) {
        for(int i = 1; i < depth - 1; i++){
            for(int j = 1; j < height - 1; j++){
                for(int k = 1; k < width - 1; k++) {
                    int index = k + j * width + i * width * height;
                    mat_tmp[index] = division * (
                    mat[index + 1]            + mat[index - 1] +
                    mat[index + width]        + mat[index - width] +
                    mat[index + width*height] + mat[index - width*height]);
                }
            }
        }

        double *mat_tmp_swap = mat_tmp;
        mat_tmp = mat;
        mat = mat_tmp_swap;

        iter--;
    }

    end = clock();

    printf("Time(event) - %.5f s\n", ((double) (end - start)) / CLOCKS_PER_SEC);

    // Creates an output which can be used to compare the different resulting grids
    if(create_matrix == 1){
        FILE *fptr;
        char filename[30];
        sprintf(filename, "matrices/CPUgrid%i_%i_%i.txt", width, height, depth);
        fptr = fopen(filename, "w");
        for(int i = 0; i < depth; i++){
            for(int j = 0; j < height; j++){
                for(int k = 0; k < width; k++){
                    fprintf(fptr, "%.16f ", mat[k + j*width + i*width*height]);
                }
            fprintf(fptr, "\n");
            }
        }
        fclose(fptr);
    }

    free(mat);
    free(mat_tmp);

    return 0;
}
