#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>

#include "programs/errorHandle.h"
#include "programs/scenarios.h"
#include <nvtx3/nvToolsExt.h>

void fillValues3D(double *mat, int width, int height, int depth, double dx, double dy, double dz) {
    double x, y, z;

    // Assuming the data in the matrix is stored contiguously in memory
    memset(mat, 0, height * width * depth * sizeof(double));


    for (int i = 1; i < depth - 1; i++) {
        z = i * dz; // z coordinate
        for (int j = 1; j < height - 1; j++) {
            y = j * dy; // z coordinate
            for (int k = 1; k < width - 1; k++) {
                x = k * dx; // x coordinate
                mat[k +  j*width + i*width*height] = sin(M_PI * x) * sin(M_PI * y) * sin(M_PI * z);
            }
        }
    }
}

void initialization(int width, int height, int depth, int iter, double dx, double dy, double dz, int gpus, int compare, int overlap, int test, dim3 blockDim, dim3 gridDim){
    /*
    Variables            | Type        | Description
    total            |    | int         | The total number of elements within the matrix
    tmp_iter             | int         | Used to remeber how many iterations we want run
    threadSize           | int         | Finds the total amount of threads in use
    gpus                 | int         | Number of gpus in use
    device_nr            | int*        | Allows the GPU to know its GPU index

    rows_total           | int         | Total number of rows to be computed on
    rows_per_device      | int         | Number of rows per device, rounded down
    rows_leftover        | int         | Number of rows leftover when rounded down

    rows_device          | int*        | Rows to allocate on the GPU
    rows_compute_device  | int*        | Rows the GPU will compute on
    rows_staring_index   | int*        | Index on the CPU matrix that the first element of the GPU matrix belongs

    threadInformation[0] | int         | Number of computations per thread on GPU 0, rounded down
    threadInformation[1] | int         | Number of computations left over when rounded down
    threadInformation[2] | int         | Number of computations per thread on GPU n-1, rounded down, is used as if there are an unequal amount of rows between device, 
                                         the first and last GPU will certainly be in each group
    threadInformation[3] | int         | Number of computations left over when rounded down
    threadInformation[4] | int         | Number of computations per thread for 1 row, rounded down
    threadInformation[5] | int         | Number of computations left over for 1 row when rounded down

    mat                  | double*     | The matrix allocated on the CPU
    mat_gpu              | double**    | One of the matrices allocated on the GPU
    mat_gpu_tmp          | double**    | The other matrix allocated on the GPU

    kernelCollEdge       | void***     | The inputfeatures to the jacobiEdge GPU kernel
    kernelCollMid        | void***     | The inputfeatures to the jacobiMid GPU kernel

    streams              | hipStream_t| The streams which is utilized when computing on the GPU
    events               | hipEvent_t | The events used to synchronize the streams
    startevent           | hipEvent_t | The event used to start the timer for the computation
    stopevent            | hipEvent_t | The event used to stop the timer for the computation

    */

   // Deler på høyde istedefor bredde?

    int total = width*height*depth;
    int threadSize = blockDim.x*blockDim.y*blockDim.z*gridDim.x*gridDim.y*gridDim.z;
    int warp_size = 32;

    int *device_nr;
    cudaErrorHandle(hipHostMalloc(&device_nr, gpus*sizeof(int*)));
    for(int g = 0; g < gpus; g++){
        device_nr[g] = g;
    }

    // Ignores first and last slice
    int slices_total = depth-2;
    int slices_per_device = slices_total/gpus;
    int slices_leftover = slices_total%gpus;
    int *slices_device, *slices_compute_device, *slices_starting_index;
    cudaErrorHandle(hipHostMalloc(&slices_device, gpus*sizeof(int*)));
    cudaErrorHandle(hipHostMalloc(&slices_starting_index, gpus*sizeof(int*)));
    cudaErrorHandle(hipHostMalloc(&slices_compute_device, gpus*sizeof(int*)));
    // Calculate the number of slices for each device
    for (int g = 0; g < gpus; g++) {
        int extra_slice = (g < slices_leftover) ? 1 : 0;
  
        slices_device[g] = slices_per_device + extra_slice + 2;

        slices_compute_device[g] = slices_per_device + extra_slice - (2*overlap); 

        slices_starting_index[g] = g * slices_per_device + min(g, slices_leftover);
    }






    // Initialiserer og allokerer Matrise på CPU
    double *mat;
    double **mat_gpu, **mat_gpu_tmp;
    cudaErrorHandle(hipHostMalloc(&mat,          total*sizeof(double)));
    cudaErrorHandle(hipHostMalloc(&mat_gpu,      gpus*sizeof(double*)));
    cudaErrorHandle(hipHostMalloc(&mat_gpu_tmp,  gpus*sizeof(double*)));

    // Allocates memory on devices based on number of rows for each device
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMalloc(&mat_gpu[g],     width*height*slices_device[g]*sizeof(double)));
        cudaErrorHandle(hipMalloc(&mat_gpu_tmp[g], width*height*slices_device[g]*sizeof(double)));
    }

    fillValues3D(mat, width, height, depth, dx, dy, dz);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMemcpy(mat_gpu[g], mat+slices_starting_index[g]*width*height, slices_device[g]*width*height*sizeof(double), hipMemcpyHostToDevice));
    }






    int *threadInformation;
    cudaErrorHandle(hipHostMalloc(&threadInformation, 6*sizeof(int)));
    threadInformation[0] = ((slices_compute_device[0])     *(width-2)*(height-2))/threadSize; // Find number of elements to compute for each thread, ignoring border elements.
    threadInformation[1] = ((slices_compute_device[0])     *(width-2)*(height-2))%threadSize; // Finding which threads require 1 more element
    threadInformation[2] = ((slices_compute_device[gpus-1])*(width-2)*(height-2))/threadSize; // Find number of elements to compute for each thread, ignoring border elements.
    threadInformation[3] = ((slices_compute_device[gpus-1])*(width-2)*(height-2))%threadSize; // Finding which threads require 1 more element
    threadInformation[4] = ((height-2)                     *(width-2))/threadSize; // Find number of elements for each thread for a slice, if 0 it means there are more threads than elements in row
    threadInformation[5] = ((height-2)                     *(width-2))%threadSize; // Finding which threads require 1 more element


    void ***kernelCollEdge;
    cudaErrorHandle(hipHostMalloc(&kernelCollEdge, gpus * sizeof(void**)));
    // Allocates the elements in the kernelCollEdge, used for hipLaunchCooperativeKernel as functon variables.
    for (int g = 0; g < gpus; g++) {
        void **kernelArgs = new void*[8];
        kernelArgs[0] = &mat_gpu[g];
        kernelArgs[1] = &mat_gpu_tmp[g];
        kernelArgs[2] = &width;
        kernelArgs[3] = &height;
        kernelArgs[4] = &slices_compute_device[g];
        kernelArgs[5] = &threadInformation[4];
        kernelArgs[6] = &threadInformation[5];
        kernelArgs[7] = &threadInformation[6];

        kernelCollEdge[g] = kernelArgs;
    }

    void ***kernelCollMid;
    cudaErrorHandle(hipHostMalloc(&kernelCollMid, gpus * sizeof(void**)));
    // Allocates the elements in the kernelCollMid, used for hipLaunchCooperativeKernel as functon variables.
    for (int g = 0; g < gpus; g++) {
        void **kernelArgs = new void*[11];
        kernelArgs[0] = &mat_gpu[g];     
        kernelArgs[1] = &mat_gpu_tmp[g];
        kernelArgs[2] = &width;
        kernelArgs[3] = &height;
        kernelArgs[4] = &slices_leftover;
        kernelArgs[5] = &device_nr[g];
        kernelArgs[6] = &slices_compute_device[g];
        kernelArgs[7] = &threadInformation[0];
        kernelArgs[8] = &threadInformation[1];
        kernelArgs[9] = &threadInformation[2];
        kernelArgs[10] = &threadInformation[3];

        kernelCollMid[g] = kernelArgs;
    }




    full_calculation_overlap(mat_gpu, mat_gpu_tmp, height, width, iter, gpus, slices_device, gridDim, blockDim, kernelCollEdge, kernelCollMid);

}



int main(int argc, char *argv[]) {
    /*
    Functions   | Type           | Input
    start       | void           | int width, int height, int iter,
                                   double dx, double dy, dim3 blockDim,
                                   dim3 gridDim

    ____________________________________________________________________________
    Variables   | Type  | Description
    width       | int   | The width of the matrix
    height      | int   | The height of the matrix

    iter        | int   | Amount of iterations
    gpus      | int   | Number of gpus in use
    compare     | int   | If one wants to compare the output with a previously CPU computed matrix
    overlap     | int   | If one want to overlap or not

    dx          | float | Used to give value to the elements of the matrix
    dy          | float | Used to give value to the elements of the matrix

    blockDim    | dim3  | Size of the threadblock
    gridDim     | dim3  | Size of the blockgrid

    For all true/false integers, 0 = false, 1 = true
    */
   
    // Checks if the correct amount of inputs is used
    if (argc != 8) {
        printf("Wrong amount of inputs: %s <width> <height> <iter> <gpus> <compare> <overlap> <test>", argv[0]);
        return 1;
    }

    int width = atoi(argv[1]);
    int height = atoi(argv[2]);
    int depth = atoi(argv[3]);
    int iter = atoi(argv[4]);
    int gpus = atoi(argv[5]);
    int compare = atoi(argv[6]);
    int overlap = atoi(argv[7]);
    int test = atoi(argv[8]);

    if(width < 1){
        printf("Width is to small\n");
    }
    else if(height < 1){
        printf("Heigth is to small\n");
    }
    else if(depth < 1){
        printf("Depth is to small\n");
    }
    else if(iter < 1){
        printf("To few selected iterations\n");
    }
    else if(gpus < 1){
        printf("Selected to few GPUs\n");
    }
    else if(compare > 1 || compare < 0){
        printf("Compare variable can only be\n"
                "0 - Do not compare the output matrix with previously created matrix\n"
                "1 - Compare with previously created matrix\n");
    }
    else if(overlap > 1 || overlap < 0){
        printf("You can only select the values\n"
                "0 - Do not overlap communication and computation\n"
                "1 - Overlap communication and computation\n");
    }
    else if(test > 4 || test < 0){
        printf("There is no test with the current value, please select one of the following.\n"
                "0 - Full Computation\n"
                "1 - No kernel\n"
                "2 - No Communication\n"
                "3 - Only computation\n"
                "4 - Only communication\n");
    }
 
    double dx = 2.0 / (width - 1);
    double dy = 2.0 / (height - 1);
    double dz = 2.0 / (depth - 1);

    dim3 blockDim(32, 32, 1);
    dim3 gridDim(16, 1, 1);

    initialization(width, height, depth, iter, dx, dy, dz, gpus, compare, overlap, test, blockDim, gridDim);

    printf("heisann");

    return 0;
}
