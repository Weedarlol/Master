#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>

#include "programs/scenarios.h"
#include "../../global_functions.h"
#include <nvtx3/nvToolsExt.h>




void initialization(int width, int height, int depth, int iter, double dx, double dy, double dz, int gpus, int compare, int overlap, int test, dim3 blockDim, dim3 gridDim){
    /*
    Variables            | Type        | Description
    total                | int         | The total number of elements within the grid
    threadSize           | int         | Finds the total amount of threads in use
    
    device_nr            | int*        | Allows the GPU to know its GPU index

    slices_total         | int         | Total number of slices to be computed on
    slices_per_device    | int         | Number of slices per device, rounded down
    slices_leftover      | int         | Number of slices leftover when rounded down

    slices_device        | int*        | slices to allocate on the GPU
    slices_compute_device| int*        | slices the GPU will compute on
    slices_staring_index | int*        | Index on the CPU grid that the first element of the GPU grid belongs

    threadInformation[0] | int         | Number of computations per thread on GPU 0, rounded down
    threadInformation[1] | int         | Number of computations left over when rounded down
    threadInformation[2] | int         | Number of computations per thread on GPU n-1, rounded down, is used as if there are an unequal amount of slices between device, 
                                         the first and last GPU will certainly be in each group
    threadInformation[3] | int         | Number of computations left over when rounded down
    threadInformation[4] | int         | Number of computations per thread for 1 slice, rounded down
    threadInformation[5] | int         | Number of computations left over for 1 slice when rounded down

    grid                  | double*     | The grid allocated on the CPU
    grid_gpu              | double**    | One of the grids allocated on the GPU
    grid_gpu_tmp          | double**    | The other grid allocated on the GPU

    kernelCollEdge       | void***     | The inputfeatures to the jacobiEdge GPU kernel
    kernelCollMid        | void***     | The inputfeatures to the jacobiMid GPU kernel
    */


    int total = width*height*depth;
    int overlap_calc = (width-2)*(height-2)*overlap;
    int threadSize = blockDim.x*blockDim.y*blockDim.z*gridDim.x*gridDim.y*gridDim.z;

    int *device_nr;
    cudaErrorHandle(hipHostMalloc(&device_nr, gpus*sizeof(int*)));
    for(int g = 0; g < gpus; g++){
        device_nr[g] = g;
    }

    // Ignores first and last slice
    int slices_total = depth-2;
    int slices_per_device = slices_total/gpus;
    int slices_leftover = slices_total%gpus;

    int *slices_device, *slices_compute_device, *slices_starting_index;
    cudaErrorHandle(hipHostMalloc(&slices_device, gpus*sizeof(int*)));
    cudaErrorHandle(hipHostMalloc(&slices_starting_index, gpus*sizeof(int*)));
    cudaErrorHandle(hipHostMalloc(&slices_compute_device, gpus*sizeof(int*)));
    // Calculate the number of slices for each device
    for (int g = 0; g < gpus; g++) {
        int extra_slice = (g < slices_leftover) ? 1 : 0;
        slices_device[g] = slices_per_device + extra_slice + 2;
        slices_compute_device[g] = slices_per_device + extra_slice - (2*overlap); // -2 as we are computing in 2 parts, 1 with point dependent on ghostpoints,and one without
        slices_starting_index[g] = g * slices_per_device + min(g, slices_leftover);
    }

    // Estimates elements per thread for 3 scenarios,
    // Scenario 1(0, 1) and 2(2, 3) are depending on if number of rows are unequal for the GPUs, which will mean that the first and last gpu has different amount of elements
    // Scenario 3(4, 5) is only used for overlap, where we only want to know the number of elements compared to threads for 1 row
    int *threadInformation;
    cudaErrorHandle(hipHostMalloc(&threadInformation, 6*sizeof(int)));
    threadInformation[0] = ((slices_compute_device[0])     *(width-2)*(height-2))/threadSize;
    threadInformation[1] = ((slices_compute_device[0])     *(width-2)*(height-2))%threadSize;
    threadInformation[2] = ((slices_compute_device[gpus-1])*(width-2)*(height-2))/threadSize;
    threadInformation[3] = ((slices_compute_device[gpus-1])*(width-2)*(height-2))%threadSize;
    threadInformation[4] = (1                              *(width-2)*(height-2))/threadSize;
    threadInformation[5] = (1                              *(width-2)*(height-2))%threadSize;

    // Initializes og allocates the grids and grid pointers on the CPU
    double *grid;
    double **grid_gpu, **grid_gpu_tmp;
    cudaErrorHandle(hipHostMalloc(&grid,          total*sizeof(double)));
    cudaErrorHandle(hipHostMalloc(&grid_gpu,      gpus*sizeof(double*)));
    cudaErrorHandle(hipHostMalloc(&grid_gpu_tmp,  gpus*sizeof(double*)));

    // Fills up the grid grid with starting values
    fillValues3D(grid, width, height, depth, dx, dy, dz);

    // Initializes og allocates the grids on the GPU
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMalloc(&grid_gpu[g],     width*height*slices_device[g]*sizeof(double)));
        cudaErrorHandle(hipMalloc(&grid_gpu_tmp[g], width*height*slices_device[g]*sizeof(double)));
    }

    void ***kernelCollEdge;
    cudaErrorHandle(hipHostMalloc(&kernelCollEdge, gpus * sizeof(void**)));
    // Allocates the elements in the kernelCollEdge, used for hipLaunchCooperativeKernel as functon variables.
    for (int g = 0; g < gpus; g++) {
        void **kernelArgs = new void*[7];
        kernelArgs[0] = &grid_gpu[g];
        kernelArgs[1] = &grid_gpu_tmp[g];
        kernelArgs[2] = &width;
        kernelArgs[3] = &height;
        kernelArgs[4] = &slices_compute_device[g];
        kernelArgs[5] = &threadInformation[4];
        kernelArgs[6] = &threadInformation[5];

        kernelCollEdge[g] = kernelArgs;
    }

    void ***kernelCollMid;
    cudaErrorHandle(hipHostMalloc(&kernelCollMid, gpus * sizeof(void**)));
    // Allocates the elements in the kernelCollMid, used for hipLaunchCooperativeKernel as functon variables.
    for (int g = 0; g < gpus; g++) {
        void **kernelArgs = new void*[13];
        kernelArgs[0] = &grid_gpu[g];
        kernelArgs[1] = &grid_gpu_tmp[g];
        kernelArgs[2] = &width;
        kernelArgs[3] = &height;
        kernelArgs[4] = &depth;
        kernelArgs[5] = &slices_leftover;
        kernelArgs[6] = &device_nr[g];
        kernelArgs[7] = &slices_compute_device[g];
        kernelArgs[8] = &threadInformation[0];
        kernelArgs[9] = &threadInformation[1];
        kernelArgs[10] = &threadInformation[2];
        kernelArgs[11] = &threadInformation[3];
        kernelArgs[12] = &overlap_calc;

        kernelCollMid[g] = kernelArgs;
    }

    // Copies elements from the grid located on the CPU to grid_gpu[g] grid located on each GPU g
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMemcpy(grid_gpu[g], grid+slices_starting_index[g]*width*height, slices_device[g]*width*height*sizeof(double), hipMemcpyHostToDevice));
    }






    full_calculation_nooverlap(grid_gpu, grid_gpu_tmp, width, height, depth, iter, gpus, slices_device, gridDim, blockDim, kernelCollMid);








    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMemcpyAsync(grid + (slices_starting_index[g]+1)*width*height, grid_gpu[g] + width*height, (slices_compute_device[g]+2*overlap)*width*height*sizeof(double), hipMemcpyDeviceToHost));
    }

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }

    // Used to compare the grid to the grid which only the CPU created
    if(compare == 1){
        double* grid_compare = (double*)malloc(width * height * depth* sizeof(double));
        FILE *fptr;
        char filename[30];
        sprintf(filename, "../CPU_3d/grids/CPUGrid%i_%i_%i.txt", width, height, depth);

        printf("Comparing the grids\n");

        fptr = fopen(filename, "r");
        if (fptr == NULL) {
            printf("Error opening file.\n");
            exit(EXIT_FAILURE);
        }

        // Read grid values from the file
        for(int i = 0; i < depth; i++){
            for (int j = 0; j < height; j++) {
                for (int k = 0; k < width; k++) {
                    if (fscanf(fptr, "%lf", &grid_compare[k + j * width + i * width * height]) != 1) {
                        printf("Error reading from file.\n");
                        fclose(fptr);
                        free(grid_compare);
                        exit(EXIT_FAILURE);
                    }
                }
            }
        }
        

        fclose(fptr);

        for(int i = 0; i < depth; i++){
            for (int j = 0; j < height; j++) {
                for (int k = 0; k < width; k++) {
                    if (fabs(grid[k + j * width + i * width * height] - grid_compare[k + j * width + i * width * height]) > 1e-15)  {
                        printf("Mismatch found at position (width = %d, height = %d, depth = %d) (grid = %.16f, grid_compare = %.16f)\n", k, j, i, grid[k + j * width + i * width * height], grid_compare[k + j * width + i * width * height]);
                        free(grid_compare);
                        exit(EXIT_FAILURE);
                    }
                }
            }
        }


        printf("All elements match!\n");
        

        // Free allocated memory
        free(grid_compare);
    }



    // Frees up memory as we are finished with the program
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipFree(grid_gpu[g]));
        cudaErrorHandle(hipFree(grid_gpu_tmp[g]));
    }
    cudaErrorHandle(hipHostFree(grid));
    cudaErrorHandle(hipHostFree(grid_gpu));
    cudaErrorHandle(hipHostFree(grid_gpu_tmp));
    cudaErrorHandle(hipHostFree(threadInformation));
    cudaErrorHandle(hipHostFree(device_nr));
    cudaErrorHandle(hipHostFree(slices_device));
    cudaErrorHandle(hipHostFree(slices_starting_index));
    cudaErrorHandle(hipHostFree(slices_compute_device));

}



int main(int argc, char *argv[]) {
    /*
    Functions   | Type           | Input
    start       | void           | int width, int height, int iter, double dx, double dy, 
                                    int gpus, int compare, int overlap, int test, 
                                    dim3 blockDim, dim3 gridDim
    ____________________________________________________________________________
    Variables   | Type  | Description
    width       | int   | The width of the grid
    height      | int   | The height of the grid
    depth       | int   | The depth of the grid
    iter        | int   | Amount of iterations
    gpus        | int   | Number of gpus in use
    compare     | int   | If one wants to compare the output with a previously CPU computed grid
    overlap     | int   | If one want to overlap or not
    test        | int   | Select which type of test one wants to run

    dx          | float | Used to give value to the elements of the grid
    dy          | float | Used to give value to the elements of the grid
    dz          | float | Used to give value to the elements of the grid

    blockDim    | dim3  | Size of the threadblock
    gridDim     | dim3  | Size of the blockgrid

    For all true/false integers, 0 = false, 1 = true
    */

    // Checks if the correct amount of inputs is used
    if (argc != 9) {
        printf("Wrong amount of inputs: %s <width> <height> <depth> <iter> <gpus> <compare> <overlap> <test>", argv[0]);
        return 1;
    }

    int width = atoi(argv[1]);
    int height = atoi(argv[2]);
    int depth = atoi(argv[3]);
    int iter = atoi(argv[4]);
    int gpus = atoi(argv[5]);
    int compare = atoi(argv[6]);
    int overlap = atoi(argv[7]);
    int test = atoi(argv[8]);

    if(width < 1){
        printf("Width is to small\n");
    }
    else if(height < 1){
        printf("Heigth is to small\n");
    }
    else if(depth < 1){
        printf("Depth is to small\n");
    }
    else if(iter < 1){
        printf("To few selected iterations\n");
    }
    else if(gpus < 1){
        printf("Selected to few GPUs\n");
    }
    else if(compare > 1 || compare < 0){
        printf("Compare variable can only be\n"
                "0 - Do not compare the output grid with previously created grid\n"
                "1 - Compare with previously created grid\n");
    }
    else if(overlap > 1 || overlap < 0){
        printf("You can only select the values\n"
                "0 - Do not overlap communication and computation\n"
                "1 - Overlap communication and computation\n");
    }
    else if(test > 4 || test < 0){
        printf("There is no test with the current value, please select one of the following.\n"
                "0 - Full Computation\n"
                "1 - No kernel\n"
                "2 - No Communication\n"
                "3 - Only computation\n"
                "4 - Only communication\n");
    }
 
    double dx = 2.0 / (width - 1);
    double dy = 2.0 / (height - 1);
    double dz = 2.0 / (depth - 1);

    dim3 blockDim(32, 32, 1);
    dim3 gridDim(16, 1, 1);

    initialization(width, height, depth, iter, dx, dy, dz, gpus, compare, overlap, test, blockDim, gridDim);

    return 0;
}
