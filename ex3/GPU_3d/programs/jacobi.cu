#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

__device__ void calc(double *data_gpu, double *data_gpu_tmp, int elementsPerThread, int index_start, int width, int height, int thread, int thread_size){
    double division = 1.0/6;
    for(int i = 0; i < elementsPerThread; i++){
        int index = index_start + i*thread_size;
        int x = index % (width - 2) + 1;
        int y = (index / (width - 2)) % (height - 2) + 1;
        int z = index / ((width - 2) * (height - 2)) + 1;
        index = x + y*width + z*width*height;

        data_gpu_tmp[index] = division * (
            data_gpu[index + 1]            + data_gpu[index - 1] +
            data_gpu[index + width]        + data_gpu[index - width] +
            data_gpu[index + width*height] + data_gpu[index - width*height]);
    }   
}

__global__ void jacobiEdge(double *data_gpu, double *data_gpu_tmp, int width, int height, 
                        int slices_compute, int elementsPerThread, int leftover){

    cg::grid_group grid_g = cg::this_grid();
    int thread = grid_g.thread_rank();
    int thread_size = grid_g.size();

    // There are more threads than elements 
    if(elementsPerThread > 0){
        if(thread < leftover){
            elementsPerThread++;
        }
        calc(data_gpu, data_gpu_tmp, elementsPerThread, thread, width, height, thread, thread_size);
        calc(data_gpu, data_gpu_tmp, elementsPerThread, thread + (slices_compute+1)*(width-2)*(height-2), width, height, thread, thread_size);
    }
    // There are less threads than elements in 1 slice
    else{
        if(thread_size >= leftover*2){
            elementsPerThread++;
            // Selects all threads with index less than width
            if(thread < leftover){
                calc(data_gpu, data_gpu_tmp, elementsPerThread, thread, width, height, thread, thread_size);
            }
            // Selects all threads with index between width and width*2
            else if(thread < leftover+leftover){
                calc(data_gpu, data_gpu_tmp, elementsPerThread, thread + (slices_compute+1)*(width-2)*(height-2), width, height, thread, thread_size);
            }
        }
        else{
            elementsPerThread++;
            if(thread < leftover){
                // The same threads will compute both slices
                calc(data_gpu, data_gpu_tmp, elementsPerThread, thread, width, height, thread, thread_size);
                calc(data_gpu, data_gpu_tmp, elementsPerThread, thread + (slices_compute+1)*(width-2)*(height-2), width, height, thread, thread_size);
            }
        }
    }
}



__global__ void jacobiMid(double *data_gpu, double *data_gpu_tmp, int width, int height,
                        int slices_Leftover, int device_nr, int slices_compute, int elementsPerThreadExtra, int elementsLeftoverExtra,
                        int elementsPerThread, int elementsLeftover, int overlap_calc){


    cg::grid_group grid_g = cg::this_grid();
    int thread = grid_g.thread_rank(); 
    int thread_size = grid_g.size();


    if(device_nr < slices_Leftover){
        if(thread < elementsLeftoverExtra){
            elementsPerThreadExtra++;
        }
        calc(data_gpu, data_gpu_tmp, elementsPerThreadExtra, thread+overlap_calc, width, height, thread, thread_size);
    }
    else{
        if(thread < elementsLeftover){
            elementsPerThread++;
        }
        calc(data_gpu, data_gpu_tmp, elementsPerThread, thread+overlap_calc, width, height, thread, thread_size);
    }
}