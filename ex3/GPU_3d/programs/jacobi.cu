#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

__device__ void calc(double *mat_gpu, double *mat_gpu_tmp, int amountPerThread, int index_start, int width, int height, int thread, int thread_size){
    double division = 1.0/6;
    for(int i = 0; i < amountPerThread; i++){
        int index = index_start + i*thread_size;
        int x = index % (width - 2) + 1;
        int y = (index / (width - 2)) % (height - 2) + 1;
        int z = index / ((width - 2) * (height - 2)) + 1;
        index = x + y*width + z*width*height;

        mat_gpu_tmp[index] = division * (
            mat_gpu[index + 1]            + mat_gpu[index - 1] +
            mat_gpu[index + width]        + mat_gpu[index - width] +
            mat_gpu[index + width*height] + mat_gpu[index - width*height]);
    }   
}

__global__ void jacobiEdge(double *mat_gpu, double *mat_gpu_tmp, int width, int height, 
                        int slices_compute, int amountPerThread, int leftover){

    cg::grid_group grid_g = cg::this_grid();
    int thread = grid_g.thread_rank();
    int thread_size = grid_g.size();

    // More threads than elements in 2 slices
    if(thread_size > leftover*2){
        amountPerThread++;
        // Selects all threads with index less than width
        if(thread < leftover){
            calc(mat_gpu, mat_gpu_tmp, amountPerThread, thread, width, height, thread, thread_size);
        }
        // Selects all threads with index between width and width*2
        else if(thread > leftover && thread < leftover+leftover){
            calc(mat_gpu, mat_gpu_tmp, amountPerThread, thread+slices_compute*(width-2), width, height, thread, thread_size);
        }
    }
    else if(thread_size > leftover){
        amountPerThread++;
        if(thread < leftover){
            // The same threads will compute both slices
            calc(mat_gpu, mat_gpu_tmp, amountPerThread, thread, width, height, thread, thread_size);
            calc(mat_gpu, mat_gpu_tmp, amountPerThread+slices_compute*(width-2), thread, width, height, thread, thread_size);
        }
    }
    // There are less threads than elements in 1 slice
    else{
        calc(mat_gpu, mat_gpu_tmp, amountPerThread, thread, width, height, thread, thread_size);
        calc(mat_gpu, mat_gpu_tmp, amountPerThread+slices_compute*(width-2), thread, width, height, thread, thread_size);
    }
}



__global__ void jacobiMid(double *mat_gpu, double *mat_gpu_tmp, int width, int height, int depth,
                        int slices_elementsLeftover, int device_nr, int slices_compute, int elementsPerThreadExtra, int elementsLeftoverExtra,
                        int elementsPerThread, int elementsLeftover, int overlap_calc){


    cg::grid_group grid_g = cg::this_grid();
    int thread = grid_g.thread_rank(); 
    int threadSize = grid_g.size();


    if(device_nr < slices_elementsLeftover){
        if(thread < elementsLeftoverExtra){
            elementsPerThreadExtra++;
        }
        calc(mat_gpu, mat_gpu_tmp, elementsPerThreadExtra, thread+overlap_calc, width, height, thread, threadSize);
    }
    else{
        if(thread < elementsLeftover){
            elementsPerThread++;
        }
        calc(mat_gpu, mat_gpu_tmp, elementsPerThread, thread+overlap_calc, width, height, thread, threadSize);
    }
}