#include <stdio.h>
#include <math.h>


#include "errorHandle.h"
#include "jacobi.h"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

void initializeStreamsAndEvents(int gpus, hipStream_t streams[][2], hipEvent_t events[][4], hipEvent_t *startevent, hipEvent_t *stopevent){
    for (int g = 0; g < gpus; g++) {
        hipSetDevice(g);
        cudaErrorHandle(hipStreamCreate(&streams[g][0]));
        cudaErrorHandle(hipStreamCreate(&streams[g][1]));
        cudaErrorHandle(hipEventCreate(&events[g][0]));
        cudaErrorHandle(hipEventCreate(&events[g][1]));
        cudaErrorHandle(hipEventCreate(&events[g][2]));
        cudaErrorHandle(hipEventCreate(&events[g][3]));
    }
    cudaErrorHandle(hipEventCreate(startevent));
    cudaErrorHandle(hipEventCreate(stopevent));
}

void freeStreamsAndEvents(int gpus, hipStream_t streams[][2], hipEvent_t events[][4], hipEvent_t *startevent, hipEvent_t *stopevent) {
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipStreamDestroy(streams[g][0]));
        cudaErrorHandle(hipStreamDestroy(streams[g][1]));
        cudaErrorHandle(hipEventDestroy(events[g][0]));
        cudaErrorHandle(hipEventDestroy(events[g][1]));
        cudaErrorHandle(hipEventDestroy(events[g][2]));
        cudaErrorHandle(hipEventDestroy(events[g][3]));
    }
}





void full_calculation_overlap(hipPitchedPtr mat_gpu, int width, int height, int depth){

}

