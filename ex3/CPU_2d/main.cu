#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>


#include "../../global_functions.h"


void fillValues(double *mat, double dx, double dy, int width, int height){
    double x, y;

    memset(mat, 0, height*width*sizeof(double));

    for(int i = 1; i < height - 1; i++) {
        y = i * dy; // y coordinate
        for(int j = 1; j < width - 1; j++) {
            x = j * dx; // x coordinate
            mat[j + i*width] = sin(M_PI*y)*sin(M_PI*x);
        }
    }
}

int main(int argc, char *argv[]) {
    /*
    width       | int   | The width of the matrix
    height      | int   | The height of the matrix
    iter        | int   | Number of max iterations for the jacobian algorithm

    eps         | double | The limit for accepting the state of the matrix during jacobian algorithm
    maxdelta    | double | The largest difference in the matrix between an iteration
    dx          | double | Distance between each element in the matrix in x direction
    dy          | double | Distance between each element in the matrix in y direction

    mat         |*double | Pointer to the matrix
    mat_tmp     |*double | Pointer to the matrix
    */

    if (argc != 4) {
        printf("Usage: %s <Width> <Height> <Iterations>", argv[0]); // Programname
        return 1;
    }

    int width = atoi(argv[1]);
    int height = atoi(argv[2]);
    int iter = atoi(argv[3]);

    double dx = 2.0 / (width - 1);
    double dy = 2.0 / (height - 1);

    double *mat;
    double *mat_tmp;

    clock_t start, end;

    mat = (double*)malloc(width*height*sizeof(double));
    mat_tmp = (double*)malloc(width*height*sizeof(double));

    /* initialization */
    fillValues(mat, dx, dy, width, height);

    start = clock();

    /* Performing Jacobian Matrix Calculation */
    // Performing a number of iterations while statement is not satisfied
    while (iter > 0) {
        // Loops through the matrix from element 1 to -2
        for(int i = 1; i < height - 1; i++){
            // Calculates the element value from row
            int i_nr = i*width;
            // Loops through the matrix from element 1 to -2
            for(int j = 1; j < width - 1; j++) {
                // Calculates each element in the matrix from itself and neightbor values.
                mat_tmp[i_nr + j] = 0.25 * (
                    mat[i_nr + j + 1]     + mat[i_nr + j - 1] +
                    mat[i_nr + j + width] + mat[i_nr + j - width]);
            }
        }

        iter--;

        /* pointer swapping */
        double *mat_tmp_cha = mat_tmp;
        mat_tmp = mat;
        mat = mat_tmp_cha;
    }

    end = clock();

    printf("Time(event) - %.5f s\n", ((double) (end - start)) / CLOCKS_PER_SEC);




    int create_matrix = 1;
    // Creates an output which can be used to compare the different resulting matrixes
    if(create_matrix == 1){
        FILE *fptr;
        char filename[30];
        sprintf(filename, "matrices/CPUMatrix%i_%i.txt", width, height);
        fptr = fopen(filename, "w");
        for(int i = 0; i < height; i++){
            for(int j = 0; j < width; j++){
                fprintf(fptr, "%.16f ", mat[j + i*width]);
            }
            fprintf(fptr, "\n");
        }
        fclose(fptr);
    }
    


    free(mat);
    free(mat_tmp);


    return 0;
}
