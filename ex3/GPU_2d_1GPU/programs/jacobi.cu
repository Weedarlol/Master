#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <nvtx3/nvToolsExt.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

__device__ void calc(double *data_gpu, double *data_gpu_tmp, int iter, int index_start, int amountPerThread,
     int thread_size, int width, int height, cg::grid_group grid_g){


    // Calculating Jacobian Matrix
    while(iter > 0){
        // Calculates each element except the border
        for(int i = 0; i < amountPerThread; i++){
            int index = index_start + i*thread_size;
            int x = index % (width - 2) + 1;
            int y = index / (height - 2) + 1;
            int ind = x + y * width;

            data_gpu_tmp[ind] = 0.25 * (
                data_gpu[ind + 1]     + data_gpu[ind - 1] +
                data_gpu[ind + width] + data_gpu[ind - width]);
        }

        // Changes pointers
        double *data_tmp_cha = data_gpu_tmp;
        data_gpu_tmp = data_gpu;
        data_gpu = data_tmp_cha;

        iter--;

        grid_g.sync();
    }
}

__global__ void jacobi(double *data_gpu, double *data_gpu_tmp, int width, int height, int iter){
    /*
    Variables      | Type      | Description
    grid_g         | grid_group| Creates a group compromising of all the threads
    thread_size     | int       | Total number of available threads within the grid_g group
    jacobiSize     | int       | Number of elements in the matrix which is to be calculated each iteration
    amountPerThread| int       | Number of elements to be calculated by each thread each iteration
    leftover       | int       | Number of threads which is required to compute one more element to be calculate all the elements
    thread         | int       | The index of each thread
    index_start    | int       | Element index the thread will start computing on, unique for each thread in grid_g group
    */

    cg::grid_group grid_g = cg::this_grid();
    int thread_size = grid_g.num_threads();
    int thread = grid_g.thread_rank();
    int jacobiSize = (width - 2) * (height - 2);
    int amountPerThread = jacobiSize / thread_size;
    int leftover = jacobiSize % thread_size;
    //int index_start = thread * amountPerThread + min(thread, leftover); //- (thread < leftover ? thread : 0);

    if(thread < leftover){
        amountPerThread++;
    }

    calc(data_gpu, data_gpu_tmp, iter, thread, amountPerThread, thread_size, width, height, grid_g);
}