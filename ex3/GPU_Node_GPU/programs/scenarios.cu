#include <stdio.h>
#include <math.h>

#include "cuda_functions.h"
#include "jacobi.h"
#include <mpi.h>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;





void full_calculation_overlap(double **data_gpu, double **data_gpu_tmp, int width, int height, int depth_node, int iter, int gpus, int rank, int size, int *slices_device, dim3 gridDim, dim3 blockDim, void*** kernelCollEdge, void*** kernelCollMid){
    /* hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
    int size = 2;
    MPI_Request myRequest[2];
    MPI_Status myStatus[2];


    double *data_cpu;
    cudaErrorHandle(hipHostMalloc(&data_cpu, width*height*2*sizeof(double)));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 1
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            // Computes the upper and lower slice
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiEdge, gridDim, blockDim, kernelCollEdge[g], 0, streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
            // Computes the rest of the slices
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
        }

        // Step 2
        // GPU to GPU Communication!
        for(int g = 1; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipStreamWaitEvent(streams[g][0], events[g][0]));
            cudaErrorHandle(hipMemcpyPeerAsync(data_gpu_tmp[g-1] + (slices_device[g-1]-1)*width*height, g-1, data_gpu_tmp[g] + width*height, g, width*height*sizeof(double), streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][2], streams[g][0]));
        }
        for(int g = 0; g < gpus-1; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipStreamWaitEvent(streams[g][0], events[g][0]));
            cudaErrorHandle(hipMemcpyPeerAsync(data_gpu_tmp[g+1], g+1, data_gpu_tmp[g] + (slices_device[g]-2)*width*height,                  g, width*height*sizeof(double), streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][3], streams[g][0]));
        }


        // CPU to CPU communication
        if(rank == 0){
            cudaErrorHandle(hipSetDevice(gpus-1));
            cudaErrorHandle(hipMemcpy(data_cpu, data_gpu_tmp[gpus-1] + (slices_device[gpus-1]-2)*width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));
        }
        else if(rank == 1){
            cudaErrorHandle(hipSetDevice(0));
            cudaErrorHandle(hipMemcpy(data_cpu, data_gpu_tmp[0] + width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));
        }
        if(rank == 0){
            MPI_Isend(&data_gpu_tmp[width*height*(depth_node-2)], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[0]);
            MPI_Irecv(&data_gpu_tmp[width*height*(depth_node-1)], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[1]); 

            MPI_Waitall(2, myRequest, myStatus);
        }
        else if(rank == size-1){
            MPI_Irecv(&data_gpu_tmp[0],                           width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[rank*2-2]); 
            MPI_Isend(&data_gpu_tmp[width*height],                width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[rank*2-1]); 

            MPI_Waitall(2, &myRequest[rank*2-2], myStatus);
        }
        else{
            MPI_Irecv(&data_gpu_tmp[0],                           width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[rank*2-2]);
            MPI_Isend(&data_gpu_tmp[width*height],                width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[rank*2-1]); 

            MPI_Isend(&data_gpu_tmp[width*height*(depth_node-2)], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[rank*2]);
            MPI_Irecv(&data_gpu_tmp[width*height*(depth_node-1)], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[rank*2+1]);

            MPI_Waitall(4, &myRequest[rank*2 - 2], myStatus);
        }
        MPI_Barrier(MPI_COMM_WORLD);
        if(rank == 0){
            cudaErrorHandle(hipSetDevice(gpus-1));
            cudaErrorHandle(hipMemcpy(data_gpu_tmp[gpus-1] + (slices_device[gpus-1]-1)*width*height, data_cpu +  width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
        }
        else if(rank == 1){
            cudaErrorHandle(hipSetDevice(0));
            cudaErrorHandle(hipMemcpy(data_gpu_tmp[0],                                                data_cpu + width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
        }

        // Step 3
        for (int g = 0; g < gpus; g++) {
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
            cudaErrorHandle(hipEventSynchronize(events[g][2]));
            cudaErrorHandle(hipEventSynchronize(events[g][3]));
        }
        
        // Step 4
        for(int g = 0; g < gpus; g++){
            double *data_change = data_gpu[g];
            data_gpu[g] = data_gpu_tmp[g];
            data_gpu_tmp[g] = data_change;
        }
        iter--;
    }

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }

    cudaErrorHandle(hipEventRecord(stopevent));
    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.5f s\n", milliseconds/1000);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent); */
}



void full_calculation_nooverlap(double **data_gpu, double **data_gpu_tmp, int width, int height, int depth_node, int iter, int gpus, int rank, int size, int *slices_device, dim3 gridDim, dim3 blockDim, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
    MPI_Request myRequest[4];
    MPI_Status myStatus[4];
    double *data_cpu;
    cudaErrorHandle(hipHostMalloc(&data_cpu, 4*width*height*sizeof(double)));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 1
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }
        
        if(rank == 0){
            for(int g = 1; g < gpus; g++){
                cudaErrorHandle(hipSetDevice(g));
                cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                cudaErrorHandle(hipMemcpyPeerAsync(data_gpu_tmp[g-1] + (slices_device[g-1]-1)*width*height, g-1, data_gpu_tmp[g] + width*height, g, width*height*sizeof(double), streams[g][1]));
                cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
            }
            // Transfers n-2 slice of the matrix
            for(int g = 0; g < gpus-1; g++){
                cudaErrorHandle(hipSetDevice(g));
                cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                cudaErrorHandle(hipMemcpyPeerAsync(data_gpu_tmp[g+1], g+1, data_gpu_tmp[g] + (slices_device[g]-2)*width*height, g, width*height*sizeof(double), streams[g][1]));
                cudaErrorHandle(hipEventRecord(events[g][2], streams[g][1]));
            }
            
            cudaErrorHandle(hipSetDevice(gpus-1));
            cudaErrorHandle(hipMemcpy(data_cpu, data_gpu_tmp[gpus-1] + (slices_device[gpus-1]-2)*width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));

            MPI_Isend(&data_cpu[0],              width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[0]);
            MPI_Irecv(&data_cpu[3*width*height], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[1]); 

            MPI_Waitall(2, myRequest, myStatus);

            cudaErrorHandle(hipMemcpy(data_gpu_tmp[gpus-1] + (slices_device[gpus-1]-1)*width*height, data_cpu +  3*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
        }
        else if(rank == size-1){
            for(int g = 1; g < gpus; g++){
                cudaErrorHandle(hipSetDevice(g));
                cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                cudaErrorHandle(hipMemcpyPeerAsync(data_gpu_tmp[g-1] + (slices_device[g-1]-1)*width*height, g-1, data_gpu_tmp[g] + width*height, g, width*height*sizeof(double), streams[g][1]));
                cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
            }
            // Transfers n-2 slice of the matrix
            for(int g = 0; g < gpus-1; g++){
                cudaErrorHandle(hipSetDevice(g));
                cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                cudaErrorHandle(hipMemcpyPeerAsync(data_gpu_tmp[g+1], g+1, data_gpu_tmp[g] + (slices_device[g]-2)*width*height, g, width*height*sizeof(double), streams[g][1]));
                cudaErrorHandle(hipEventRecord(events[g][2], streams[g][1]));
            }
            
            cudaErrorHandle(hipSetDevice(0));
            cudaErrorHandle(hipMemcpy(data_cpu + width*height, data_gpu_tmp[0] + width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));

            MPI_Isend(&data_cpu[width*height], width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[0]);
            MPI_Irecv(&data_cpu[2*width*height], width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[1]); 

            MPI_Waitall(2, myRequest, myStatus);

            cudaErrorHandle(hipMemcpy(data_gpu_tmp[gpus-1], data_cpu +  2*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
        }
        else{
            for(int g = 1; g < gpus; g++){
                cudaErrorHandle(hipSetDevice(g));
                cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                cudaErrorHandle(hipMemcpyPeerAsync(data_gpu_tmp[g-1] + (slices_device[g-1]-1)*width*height, g-1, data_gpu_tmp[g] + width*height, g, width*height*sizeof(double), streams[g][1]));
                cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
            }
            // Transfers n-2 slice of the matrix
            for(int g = 0; g < gpus-1; g++){
                cudaErrorHandle(hipSetDevice(g));
                cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                cudaErrorHandle(hipMemcpyPeerAsync(data_gpu_tmp[g+1], g+1, data_gpu_tmp[g] + (slices_device[g]-2)*width*height, g, width*height*sizeof(double), streams[g][1]));
                cudaErrorHandle(hipEventRecord(events[g][2], streams[g][1]));
            }

            cudaErrorHandle(hipSetDevice(gpus-1));
            cudaErrorHandle(hipMemcpy(data_cpu, data_gpu_tmp[gpus-1] + (slices_device[gpus-1]-2)*width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));
            cudaErrorHandle(hipSetDevice(0));
            cudaErrorHandle(hipMemcpy(data_cpu + width*height, data_gpu_tmp[0] + width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));

            MPI_Irecv(&data_cpu[2*width*height], width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[0]); 
            MPI_Isend(&data_cpu[width*height],   width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[1]); 

            MPI_Isend(&data_cpu[0],              width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[2]);
            MPI_Irecv(&data_cpu[3*width*height], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[3]); 

            MPI_Waitall(4, myRequest, myStatus);

            cudaErrorHandle(hipSetDevice(gpus-1));
            cudaErrorHandle(hipMemcpy(data_gpu_tmp[gpus-1] + (slices_device[gpus-1]-1)*width*height, data_cpu +  3*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
            cudaErrorHandle(hipSetDevice(0));
            cudaErrorHandle(hipMemcpy(data_gpu_tmp[gpus-1], data_cpu +  2*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));

        }

        // Step 3
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
            cudaErrorHandle(hipEventSynchronize(events[g][2]));
        }
        
        // Step 4
        for(int g = 0; g < gpus; g++){
            double *data_change = data_gpu[g];
            data_gpu[g] = data_gpu_tmp[g];
            data_gpu_tmp[g] = data_change;
        iter--;
    }

    
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.5f s\n", milliseconds/1000);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}

/* 
void full_calculation_nooverlap(double **data_gpu, double **data_gpu_tmp, int width, int height, int depth, int iter, int gpus, int rank, int size, int *slices_device, dim3 gridDim, dim3 blockDim, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
    MPI_Request myRequest[4];
    MPI_Status myStatus[4];
    double *data_cpu;
    cudaErrorHandle(hipHostMalloc(&data_cpu, 4*width*height*sizeof(double)));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 1
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }


        if(rank == 0){
            for(int g = 1; g < gpus; g++){
                cudaErrorHandle(hipSetDevice(g));
                cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                cudaErrorHandle(hipMemcpyPeerAsync(data_gpu_tmp[g-1] + (slices_device[g-1]-1)*width*height, g-1, data_gpu_tmp[g] + width*height, g, width*height*sizeof(double), streams[g][1]));
                cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
            }
            // Transfers n-2 slice of the matrix
            for(int g = 0; g < gpus-1; g++){
                cudaErrorHandle(hipSetDevice(g));
                cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                cudaErrorHandle(hipMemcpyPeerAsync(data_gpu_tmp[g+1], g+1, data_gpu_tmp[g] + (slices_device[g]-2)*width*height, g, width*height*sizeof(double), streams[g][1]));
                cudaErrorHandle(hipEventRecord(events[g][2], streams[g][1]));
            }
            
            cudaErrorHandle(hipSetDevice(gpus-1));
            cudaErrorHandle(hipMemcpy(data_cpu, data_gpu_tmp[gpus-1] + (slices_device[gpus-1]-2)*width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));

            MPI_Isend(&data_cpu[0],              width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[0]);
            MPI_Irecv(&data_cpu[3*width*height], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[1]); 

            MPI_Waitall(2, myRequest, myStatus);

            cudaErrorHandle(hipMemcpy(data_gpu_tmp[gpus-1] + (slices_device[gpus-1]-1)*width*height, data_cpu +  3*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
        }
        else if(rank == size-1){
            for(int g = 1; g < gpus; g++){
                cudaErrorHandle(hipSetDevice(g));
                cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                cudaErrorHandle(hipMemcpyPeerAsync(data_gpu_tmp[g-1] + (slices_device[g-1]-1)*width*height, g-1, data_gpu_tmp[g] + width*height, g, width*height*sizeof(double), streams[g][1]));
                cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
            }
            // Transfers n-2 slice of the matrix
            for(int g = 0; g < gpus-1; g++){
                cudaErrorHandle(hipSetDevice(g));
                cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                cudaErrorHandle(hipMemcpyPeerAsync(data_gpu_tmp[g+1], g+1, data_gpu_tmp[g] + (slices_device[g]-2)*width*height, g, width*height*sizeof(double), streams[g][1]));
                cudaErrorHandle(hipEventRecord(events[g][2], streams[g][1]));
            }
            
            cudaErrorHandle(hipSetDevice(0));
            cudaErrorHandle(hipMemcpy(data_cpu + width*height, data_gpu_tmp[0] + width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));

            MPI_Isend(&data_cpu[width*height], width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[0]);
            MPI_Irecv(&data_cpu[2*width*height], width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[1]); 

            MPI_Waitall(2, myRequest, myStatus);

            cudaErrorHandle(hipMemcpy(data_gpu_tmp[gpus-1], data_cpu +  2*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
        }
        else{
            for(int g = 1; g < gpus; g++){
                cudaErrorHandle(hipSetDevice(g));
                cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                cudaErrorHandle(hipMemcpyPeerAsync(data_gpu_tmp[g-1] + (slices_device[g-1]-1)*width*height, g-1, data_gpu_tmp[g] + width*height, g, width*height*sizeof(double), streams[g][1]));
                cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
            }
            // Transfers n-2 slice of the matrix
            for(int g = 0; g < gpus-1; g++){
                cudaErrorHandle(hipSetDevice(g));
                cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                cudaErrorHandle(hipMemcpyPeerAsync(data_gpu_tmp[g+1], g+1, data_gpu_tmp[g] + (slices_device[g]-2)*width*height, g, width*height*sizeof(double), streams[g][1]));
                cudaErrorHandle(hipEventRecord(events[g][2], streams[g][1]));
            }

            cudaErrorHandle(hipSetDevice(gpus-1));
            cudaErrorHandle(hipMemcpy(data_cpu, data_gpu_tmp[gpus-1] + (slices_device[gpus-1]-2)*width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));
            cudaErrorHandle(hipSetDevice(0));
            cudaErrorHandle(hipMemcpy(data_cpu + width*height, data_gpu_tmp[0] + width*height, width*height*sizeof(double), hipMemcpyDeviceToHost));

            MPI_Irecv(&data_cpu[2*width*height], width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[0]); 
            MPI_Isend(&data_cpu[width*height],   width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[1]); 

            MPI_Isend(&data_cpu[0],              width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[2]);
            MPI_Irecv(&data_cpu[3*width*height], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[3]); 

            MPI_Waitall(4, myRequest, myStatus);

            cudaErrorHandle(hipSetDevice(gpus-1));
            cudaErrorHandle(hipMemcpy(data_gpu_tmp[gpus-1] + (slices_device[gpus-1]-1)*width*height, data_cpu +  3*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));
            cudaErrorHandle(hipSetDevice(0));
            cudaErrorHandle(hipMemcpy(data_gpu_tmp[gpus-1], data_cpu +  2*width*height, width*height*sizeof(double), hipMemcpyHostToDevice));

        }

        // Step 3
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
            cudaErrorHandle(hipEventSynchronize(events[g][2]));
        }
        
        // Step 4
        for(int g = 0; g < gpus; g++){
            double *data_change = data_gpu[g];
            data_gpu[g] = data_gpu_tmp[g];
            data_gpu_tmp[g] = data_change;

            void *temp_mid = kernelMid[g][0];
            kernelMid[g][0] = kernelMid[g][1];
            kernelMid[g][1] = temp_mid;
        }
        
        
        iter--;
    }
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.5f s\n", milliseconds/1000);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
} */