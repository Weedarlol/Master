#include "cuda_functions.h"


void initializeStreamsAndEventsOne(int gpus, hipStream_t streams[], hipEvent_t events[][4], hipEvent_t *startevent, hipEvent_t *stopevent) {
    for (int g = 0; g < gpus; g++) {
        hipSetDevice(g);
        cudaErrorHandle(hipStreamCreate(&streams[g]));
        cudaErrorHandle(hipEventCreate(&events[g][0]));
        cudaErrorHandle(hipEventCreate(&events[g][1]));
        cudaErrorHandle(hipEventCreate(&events[g][2]));
        cudaErrorHandle(hipEventCreate(&events[g][3]));
    }
    cudaErrorHandle(hipEventCreate(startevent));
    cudaErrorHandle(hipEventCreate(stopevent));
}

void initializeStreamsAndEventsTwo(int gpus, hipStream_t streams[][2], hipEvent_t events[][4], hipEvent_t *startevent, hipEvent_t *stopevent) {
    for (int g = 0; g < gpus; g++) {
        hipSetDevice(g);
        cudaErrorHandle(hipStreamCreate(&streams[g][0]));
        cudaErrorHandle(hipStreamCreate(&streams[g][1]));
        cudaErrorHandle(hipEventCreate(&events[g][0]));
        cudaErrorHandle(hipEventCreate(&events[g][1]));
        cudaErrorHandle(hipEventCreate(&events[g][2]));
        cudaErrorHandle(hipEventCreate(&events[g][3]));
    }
    cudaErrorHandle(hipEventCreate(startevent));
    cudaErrorHandle(hipEventCreate(stopevent));
}

void freeStreamsAndEventsOne(int gpus, hipStream_t streams[], hipEvent_t events[][4], hipEvent_t *startevent, hipEvent_t *stopevent) {
    for (int g = 0; g < gpus; g++) {
        cudaErrorHandle(hipStreamDestroy(streams[g]));
        cudaErrorHandle(hipEventDestroy(events[g][0]));
        cudaErrorHandle(hipEventDestroy(events[g][1]));
        cudaErrorHandle(hipEventDestroy(events[g][2]));
        cudaErrorHandle(hipEventDestroy(events[g][3]));
    }
}

void freeStreamsAndEventsTwo(int gpus, hipStream_t streams[][2], hipEvent_t events[][4], hipEvent_t *startevent, hipEvent_t *stopevent) {
    for (int g = 0; g < gpus; g++) {
        cudaErrorHandle(hipStreamDestroy(streams[g][0]));
        cudaErrorHandle(hipStreamDestroy(streams[g][1]));
        cudaErrorHandle(hipEventDestroy(events[g][0]));
        cudaErrorHandle(hipEventDestroy(events[g][1]));
        cudaErrorHandle(hipEventDestroy(events[g][2]));
        cudaErrorHandle(hipEventDestroy(events[g][3]));
    }
}
