#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>

#include "jacobi.h"
#include <nvtx3/nvToolsExt.h>

// https://ori-cohen.medium.com/real-life-cuda-programming-part-4-error-checking-e66dcbad6b55
#define cudaErrorHandle(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) 
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort)
            exit(code);
    }
}

void fillValues(double *mat, double dx, double dy, int width, int height){
    double x, y;

    memset(mat, 0, height*width*sizeof(double));

    for(int i = 1; i < height - 1; i++) {
        y = i * dy; // y coordinate
        for(int j = 1; j < width - 1; j++) {
            x = j * dx; // x coordinate
            mat[j + i*width] = sin(M_PI*y)*sin(M_PI*x);
        }
    }
}

void start(int width, int height, int iter, double dx, double dy, int gpu_nr, int compare, int overlap, dim3 blockDim, dim3 gridDim){
    /*
    Variables            | Type        | Description
    total                | int         | The total number of elements within the matrix
    tmp_iter             | int         | Used to remeber how many iterations we want run
    overlap_calc         | int         | Used to find how many elements less the kernelCollMid has to compute when we have overlap
    threadSize           | int         | Finds the total amount of threads in use
    gpus                 | int         | Number of GPUs in use
    device_nr            | int*        | Allows the GPU to know its GPU index

    rows_total           | int         | Total number of rows to be computed on
    rows_per_device      | int         | Number of rows per device, rounded down
    rows_leftover        | int         | Number of rows leftover when rounded down

    rows_device          | int*        | Rows to allocate on the GPU
    rows_compute_device  | int*        | Rows the GPU will compute on
    rows_staring_index   | int*        | Index on the CPU matrix that the first element of the GPU matrix belongs

    threadInformation[0] | int         | Number of computations per thread on GPU 0, rounded down
    threadInformation[1] | int         | Number of computations left over when rounded down
    threadInformation[2] | int         | Number of computations per thread on GPU n-1, rounded down, is used as if there are an unequal amount of rows between device, 
                                         the first and last GPU will certainly be in each group
    threadInformation[3] | int         | Number of computations left over when rounded down
    threadInformation[4] | int         | Number of computations per thread for 1 row, rounded down
    threadInformation[5] | int         | Number of computations left over for 1 row when rounded down

    mat                  | double*     | The matrix allocated on the CPU
    mat_gpu              | double**    | One of the matrices allocated on the GPU
    mat_gpu_tmp          | double**    | The other matrix allocated on the GPU

    kernelCollEdge       | void***     | The inputfeatures to the jacobiEdge GPU kernel
    kernelCollMid        | void***     | The inputfeatures to the jacobiMid GPU kernel

    streams              | hipStream_t| The streams which is utilized when computing on the GPU
    events               | hipEvent_t | The events used to synchronize the streams
    startevent           | hipEvent_t | The event used to start the timer for the computation
    stopevent            | hipEvent_t | The event used to stop the timer for the computation

    */


    int total = width*height;
    int tmp_iter = iter;
    int overlap_calc = (width-2)*overlap;
    int threadSize = blockDim.x*blockDim.y*blockDim.z*gridDim.x*gridDim.y*gridDim.z;

    int gpus;
    int *device_nr;
    cudaErrorHandle(hipGetDeviceCount(&gpus));
    cudaErrorHandle(hipHostMalloc(&device_nr, gpus*sizeof(int*)));
    for(int g = 0; g < gpus; g++){
        device_nr[g] = g;
    }

    // Ignores first and last row
    int rows_total = height-2;
    int rows_per_device = rows_total/gpus;
    int rows_leftover = rows_total%gpus;
    int *rows_device, *rows_compute_device, *rows_starting_index;
    cudaErrorHandle(hipHostMalloc(&rows_device, gpus*sizeof(int*)));
    cudaErrorHandle(hipHostMalloc(&rows_starting_index, gpus*sizeof(int*)));
    cudaErrorHandle(hipHostMalloc(&rows_compute_device, gpus*sizeof(int*)));
    // Calculate the number of rows for each device
    for (int g = 0; g < gpus; g++) {
        int extra_row = (g < rows_leftover) ? 1 : 0;
  
        rows_device[g] = rows_per_device + extra_row + 2;

        rows_compute_device[g] = rows_per_device + extra_row - (2*overlap); // -2 as we are computing in 2 parts, 1 with point dependent on ghostpoints,and one without

        rows_starting_index[g] = g * rows_per_device + min(g, rows_leftover);
    }

    int *threadInformation;
    cudaErrorHandle(hipHostMalloc(&threadInformation, 6*sizeof(int)));
    threadInformation[0] = ((rows_compute_device[0])     *(width-2))/threadSize; // Find number of elements to compute for each thread, ignoring border elements.
    threadInformation[1] = ((rows_compute_device[0])     *(width-2))%threadSize; // Finding which threads require 1 more element
    threadInformation[2] = ((rows_compute_device[gpus-1])*(width-2))/threadSize; // Find number of elements to compute for each thread, ignoring border elements.
    threadInformation[3] = ((rows_compute_device[gpus-1])*(width-2))%threadSize; // Finding which threads require 1 more element
    threadInformation[4] = (1                            *(width-2))/threadSize; // Find number of elements for each thread for a row, if 0 it means there are more threads than elements in row
    threadInformation[5] = (1                            *(width-2))%threadSize; // Finding which threads require 1 more element

    

    double *mat;
    double **mat_gpu, **mat_gpu_tmp;
    cudaErrorHandle(hipHostMalloc(&mat,          total*sizeof(double)));
    cudaErrorHandle(hipHostMalloc(&mat_gpu,      gpus*sizeof(double*)));
    cudaErrorHandle(hipHostMalloc(&mat_gpu_tmp,  gpus*sizeof(double*)));


    // Allocates memory on devices based on number of rows for each device
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMalloc(&mat_gpu[g],     width*rows_device[g]*sizeof(double)));
        cudaErrorHandle(hipMalloc(&mat_gpu_tmp[g], width*rows_device[g]*sizeof(double)));
    }

    void ***kernelCollEdge;
    cudaErrorHandle(hipHostMalloc(&kernelCollEdge, gpus * sizeof(void**)));
    // Allocates the elements in the kernelCollEdge, used for hipLaunchCooperativeKernel as functon variables.
    for (int g = 0; g < gpus; g++) {
        void **kernelArgs = new void*[7];
        kernelArgs[0] = &mat_gpu[g];
        kernelArgs[1] = &mat_gpu_tmp[g];
        kernelArgs[2] = &width;
        kernelArgs[3] = &height;
        kernelArgs[4] = &rows_compute_device[g];
        kernelArgs[5] = &threadInformation[4];
        kernelArgs[6] = &threadInformation[5];

        kernelCollEdge[g] = kernelArgs;
    }

    void ***kernelCollMid;
    cudaErrorHandle(hipHostMalloc(&kernelCollMid, gpus * sizeof(void**)));
    // Allocates the elements in the kernelCollMid, used for hipLaunchCooperativeKernel as functon variables.
    for (int g = 0; g < gpus; g++) {
        void **kernelArgs = new void*[12];
        kernelArgs[0] = &mat_gpu[g];     
        kernelArgs[1] = &mat_gpu_tmp[g];
        kernelArgs[2] = &width;
        kernelArgs[3] = &height;
        kernelArgs[4] = &rows_leftover;
        kernelArgs[5] = &device_nr[g];
        kernelArgs[6] = &rows_compute_device[g];
        kernelArgs[7] = &threadInformation[0];
        kernelArgs[8] = &threadInformation[1];
        kernelArgs[9] = &threadInformation[2];
        kernelArgs[10] = &threadInformation[3];
        kernelArgs[11] = &overlap_calc;

        kernelCollMid[g] = kernelArgs;
    }

    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipStreamCreate(&streams[g][0]));
        cudaErrorHandle(hipStreamCreate(&streams[g][1]));
        cudaErrorHandle(hipEventCreate(&events[g][0]));
        cudaErrorHandle(hipEventCreate(&events[g][1]));
        cudaErrorHandle(hipEventCreate(&events[g][2]));
        cudaErrorHandle(hipEventCreate(&events[g][3]));
    }
    cudaErrorHandle(hipEventCreate(&startevent));
    cudaErrorHandle(hipEventCreate(&stopevent));


    fillValues(mat, dx, dy, width, height);


    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMemcpyAsync(mat_gpu[g], mat+rows_starting_index[g]*width, rows_device[g]*width*sizeof(double), hipMemcpyHostToDevice, streams[g][0]));
        cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
    }

    
    for (int g = 0; g < gpus; g++) {
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipStreamSynchronize(streams[g][0]));
        cudaErrorHandle(hipStreamSynchronize(streams[g][1]));
        cudaErrorHandle(hipEventSynchronize(events[g][0]));
        cudaErrorHandle(hipEventSynchronize(events[g][1]));
        cudaErrorHandle(hipEventSynchronize(events[g][2]));
        cudaErrorHandle(hipEventSynchronize(events[g][3]));
    }

    

    nvtxRangePushA("Area of Interest");
    cudaErrorHandle(hipEventRecord(startevent));
    
    if(overlap == 1){
        if(gpus > 1){
            while(iter > 0){
                // Step 1
                for(int g = 0; g < gpus; g++){
                    cudaErrorHandle(hipSetDevice(g));
                    // Computes the 2 rows
                    cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiEdge, gridDim, blockDim, kernelCollEdge[g], 0, streams[g][1]));
                    cudaErrorHandle(hipEventRecord(events[g][0], streams[g][1]));
                    // Computes the rest of the rows
                    cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][0]));
                    cudaErrorHandle(hipEventRecord(events[g][1], streams[g][0]));
                }



                // Step 2
                // Transfer 2 row of the matrix
                for(int g = 1; g < gpus; g++){
                    cudaErrorHandle(hipSetDevice(g));
                    cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                    cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (rows_device[g-1]-1)*width + 1, g-1, mat_gpu_tmp[g] + width + 1, g, (width-2)*sizeof(double), streams[g][1]));
                    cudaErrorHandle(hipEventRecord(events[g][2], streams[g][1]));
                }
                // Transfers n-2 row of the matrix
                for(int g = 0; g < gpus-1; g++){
                    cudaErrorHandle(hipSetDevice(g));
                    cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                    cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + 1, g+1, mat_gpu_tmp[g] + (rows_device[g]-2)*width + 1, g, (width-2)*sizeof(double), streams[g][1]));
                    cudaErrorHandle(hipEventRecord(events[g][3], streams[g][1]));
                }


                // Step 3
                for (int g = 0; g < gpus; g++) {
                    cudaErrorHandle(hipSetDevice(g));
                    cudaErrorHandle(hipEventSynchronize(events[g][1]));
                    cudaErrorHandle(hipEventSynchronize(events[g][2]));
                    cudaErrorHandle(hipEventSynchronize(events[g][3]));
                }
                
                // Step 4
                for(int g = 0; g < gpus; g++){
                    double *mat_change = mat_gpu[g];
                    mat_gpu[g] = mat_gpu_tmp[g];
                    mat_gpu_tmp[g] = mat_change;
                }
                iter--;
            }
            
        }
    }
    else{
        if(gpus > 1){
            while(iter > 0){
                // Step 1
                // Computes the 2 row
                for(int g = 0; g < gpus; g++){
                    cudaErrorHandle(hipSetDevice(g));
                    cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][0]));
                    cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
                }


                // Step 2
                // Transfers 2 row of the matrix
                for(int g = 1; g < gpus; g++){
                    cudaErrorHandle(hipSetDevice(g));
                    cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                    cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (rows_device[g-1]-1)*width + 1, g-1, mat_gpu_tmp[g] + width + 1, g, (width-2)*sizeof(double), streams[g][1]));
                    cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
                }
                // Transfers n-2 row of the matrix
                for(int g = 0; g < gpus-1; g++){
                    cudaErrorHandle(hipSetDevice(g));
                    cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
                    cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + 1, g+1, mat_gpu_tmp[g] + (rows_device[g]-2)*width + 1, g, (width-2)*sizeof(double), streams[g][1]));
                    cudaErrorHandle(hipEventRecord(events[g][2], streams[g][1]));
                }


                // Step 3
                for(int g = 0; g < gpus; g++){
                    cudaErrorHandle(hipSetDevice(g));
                    cudaErrorHandle(hipEventSynchronize(events[g][0]));
                    cudaErrorHandle(hipEventSynchronize(events[g][1]));
                    cudaErrorHandle(hipEventSynchronize(events[g][2]));
                }
                
                // Step 5
                for(int g = 0; g < gpus; g++){
                    double *mat_change = mat_gpu[g];
                    mat_gpu[g] = mat_gpu_tmp[g];
                    mat_gpu_tmp[g] = mat_change;
                }
                iter--;
            }
            
        }
    }
    
    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));
    
    nvtxRangePop();


    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", tmp_iter - iter);



    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMemcpyAsync(mat + (rows_starting_index[g]+1)*width, mat_gpu[g] + width, (rows_compute_device[g]+2*overlap)*width*sizeof(double), hipMemcpyDeviceToHost));
    }
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }


    // Used to compare the matrix to the matrix which only the CPU created
    if(compare == 1){
        double* mat_compare = (double*)malloc(width * height * sizeof(double));
        FILE *fptr;
        char filename[30];
        sprintf(filename, "../CPU/CPUMatrix%i_%i.txt", width, height);

        printf("Comparing the matrixes\n");

        fptr = fopen(filename, "r");
        if (fptr == NULL) {
            printf("Error opening file.\n");
            exit(EXIT_FAILURE);
        }

        // Read matrix values from the file
        for (int i = 0; i < height; i++) {
            for (int j = 0; j < width; j++) {
                if (fscanf(fptr, "%lf", &mat_compare[j + i * width]) != 1) {
                    printf("Error reading from file.\n");
                    fclose(fptr);
                    free(mat_compare);
                    exit(EXIT_FAILURE);
                }
            }
        }

        fclose(fptr);


        // Comparing the elements
        for (int i = 1; i < height-1; i++) {
            for (int j = 1; j < width-1; j++) {
                if (fabs(mat[j + i * width] - mat_compare[j + i * width]) > 1e-16)  {
                    printf("Mismatch found at position (%d, %d) (%.16f, %.16f)\n", i, j, mat[j + i * width], mat_compare[j + i * width]);
                    free(mat_compare);
                    exit(EXIT_FAILURE);
                    cudaErrorHandle(hipDeviceSynchronize());
                }
            }
        }


        printf("All elements match!\n");
        

        // Free allocated memory
        free(mat_compare);
    }



    // Frees up memory as we are finished with the program
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipFree(mat_gpu[g]));
        cudaErrorHandle(hipFree(mat_gpu_tmp[g]));
        cudaErrorHandle(hipStreamDestroy(streams[g][0]));
        cudaErrorHandle(hipStreamDestroy(streams[g][1]));
    }
    cudaErrorHandle(hipHostFree(mat));
    cudaErrorHandle(hipHostFree(mat_gpu));
    cudaErrorHandle(hipHostFree(mat_gpu_tmp));
    cudaErrorHandle(hipHostFree(threadInformation));
    cudaErrorHandle(hipHostFree(device_nr));
    cudaErrorHandle(hipHostFree(rows_device));
    cudaErrorHandle(hipHostFree(rows_starting_index));
    cudaErrorHandle(hipHostFree(rows_compute_device));
    // kernelCollMid and kernelCollEdge?
}



int main(int argc, char *argv[]) {
    /*
    Functions   | Type           | Input
    start       | void           | int width, int height, int iter,
                                   double dx, double dy, dim3 blockDim,
                                   dim3 gridDim

    ____________________________________________________________________________
    Variables   | Type  | Description
    width       | int   | The width of the matrix
    height      | int   | The height of the matrix
    iter        | int   | Amount of iterations
    gpu_nr      | int   | Number of GPUs in use
    compare     | int   | If one wants to compare the output with a previously CPU computed matrix
    overlap     | int   | If one want to overlap or not
    dx          | float | Used to give value to the elements of the matrix
    dy          | float | Used to give value to the elements of the matrix
    blockDim    | dim3  | Size of the threadblock
    gridDim     | dim3  | Size of the blockgrid

    For all true/false integers, 0 = false, 1 = true
    */
    // Checks if the correct amount of inputs is used
    if (argc != 7) {
        printf("Usage: %s <Width> <Height> <Iterations>", argv[0]);
        return 1;
    }

    int width = atoi(argv[1]);
    int height = atoi(argv[2]);
    int iter = atoi(argv[3]);
    int gpu_nr = atoi(argv[4]);
    int compare = atoi(argv[5]);
    int overlap = atoi(argv[6]);

    double dx = 2.0 / (width - 1);
    double dy = 2.0 / (height - 1);

    dim3 blockDim(32, 32, 1);
    dim3 gridDim(16, 1, 1);

    start(width, height, iter, dx, dy, gpu_nr, compare, overlap, blockDim, gridDim);

    return 0;
}
