#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>

#include "jacobi.h"
#include <nvtx3/nvToolsExt.h>

// https://ori-cohen.medium.com/real-life-cuda-programming-part-4-error-checking-e66dcbad6b55
#define cudaErrorHandle(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) 
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort)
            exit(code);
    }
}

void fillValues(float *mat, float dx, float dy, int width, int height){
    float x, y;

    memset(mat, 0, height*width*sizeof(float));

    for(int i = 1; i < height - 1; i++) {
        y = i * dy; // y coordinate
        for(int j = 1; j < width - 1; j++) {
            x = j * dx; // x coordinate
            mat[j + i*width] = sin(M_PI*y)*sin(M_PI*x);
        }
    }
}

void start(int width, int height, int iter, float eps, float dx, float dy, dim3 blockDim, dim3 gridDim){
    /*
    Variables   | Type  | Description
    gpus        | int   | Number of available gpus
    total       | int   | Total number of elements in the matrix
    dataPerGpu  | int   | Number of elements given to each GPU
    maxThreads  | int   | Total number of available threads within the grid_g group
    jacobiSize  | int   | Number of elements in the matrix which is to be calculated each iteration
    amountPerThread|int | Number of elements to be calculated by each thread each iteration
    leftover    | int   | Number of threads which is required to compute one more element to calculate all the elements

    start       |clock_t| Start timer of area of interest
    end         |clock_t| End timer of area of interest

    mat         |*float | Pointer to the matrix allocated in the CPU
    mat_gpu     |**float| An array of pointers, where each pointer points at an device, specifically a matrix within that device
    mat_gpu_tmp |**float| An array of pointers, where each pointer points at an device, specifically a matrix within that device
    maxEps      |**int  | An array of pointers, where each pointer points at an device, specifically an array within that device that checks if the elements is in an acceptable state
    maxEps_print|*int   | Variable used to for the CPU to check if the GPUs are finished
    device_nr   |*int   | An array used to send the GPU number to each device when computing
    */
    
    int gpus;
    cudaErrorHandle(hipGetDeviceCount(&gpus));

    int total = width*height;
    int print_iter = iter;
    int dataPerGpu = (width*height)/gpus;
    int dataLeftover = (width*height)%gpus;
    int maxThreads = blockDim.x*blockDim.y*blockDim.z*gridDim.x*gridDim.y*gridDim.z;
    int jacobiSize = ((width - 2) * (height - 2))/gpus;
    int amountPerThread = jacobiSize / maxThreads;
    int leftover = jacobiSize % maxThreads;

    clock_t start, end;

    float *mat;
    float **mat_gpu, **mat_gpu_tmp;

    int *device_nr;
    int **maxEps, *maxEps_print;

    cudaErrorHandle(hipHostMalloc(&mat, total*sizeof(float*)));

    cudaErrorHandle(hipHostMalloc(&mat_gpu,      gpus*sizeof(float*)));
    cudaErrorHandle(hipHostMalloc(&mat_gpu_tmp,  gpus*sizeof(float*)));
    cudaErrorHandle(hipHostMalloc(&device_nr,    gpus*sizeof(int*)));
    cudaErrorHandle(hipHostMalloc(&maxEps,       gpus*sizeof(int*)));
    cudaErrorHandle(hipHostMalloc(&maxEps_print, gpus*sizeof(int)));
    

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMalloc(&mat_gpu[g],     total*sizeof(float)));
        cudaErrorHandle(hipMalloc(&mat_gpu_tmp[g], total*sizeof(float)));
        cudaErrorHandle(hipMalloc(&maxEps[g],      maxThreads*sizeof(int*)));
        
        maxEps_print[g] = 1;
        device_nr[g] = g;
    }

    hipStream_t streams[gpus];
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipStreamCreate(&streams[g]));
    }

    fillValues(mat, dx, dy, width, height);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMemcpyAsync(mat_gpu[g], mat, total*sizeof(float), hipMemcpyHostToDevice, streams[g]));
    }

    void ***kernelColl;
    cudaErrorHandle(hipHostMalloc(&kernelColl, gpus * sizeof(void**)));
    // Allocates the elements in the kernelColl, used for hipLaunchCooperativeKernel as functon variables.
    for (int g = 0; g < gpus; g++) {
        void **kernelArgs = new void*[12];
        kernelArgs[0] = &mat_gpu[g];
        kernelArgs[1] = &mat_gpu_tmp[g];
        kernelArgs[2] = &maxEps[g];
        kernelArgs[3] = &device_nr[g];
        kernelArgs[4] = &dataLeftover;
        kernelArgs[5] = &eps;
        kernelArgs[6] = &width;
        kernelArgs[7] = &height;
        kernelArgs[8] = &iter;
        kernelArgs[9] = &jacobiSize;
        kernelArgs[10] = &amountPerThread;
        kernelArgs[11] = &leftover;
        
        kernelColl[g] = kernelArgs;
    }

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipStreamSynchronize(streams[g]));
    }


    start = clock();
    // ________________________________________________________
    while(iter > 0 && maxEps_print[0] != 0){
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobi, gridDim, blockDim, kernelColl[g], 0, streams[g]));
            cudaErrorHandle(hipMemcpyAsync(&maxEps_print[g], maxEps[g], sizeof(int), hipMemcpyDeviceToHost, streams[g]));
        }

        if(gpus > 1){
            for(int g = 0; g < dataLeftover; g++){
                cudaErrorHandle(hipSetDevice(g)); // Unnecessary?
                if(g == 0){
                    // Transfers data device 0 -> device 1
                    cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[1] + dataPerGpu+1-width, 1, mat_gpu_tmp[0] + dataPerGpu+1-width, 0, width*sizeof(float), streams[g]));
                }
                else if(g < gpus-1){
                    // Transfers data device g -> device g+1
                    cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + dataPerGpu*g+1-width, g+1, mat_gpu_tmp[g] + dataPerGpu*g+1-width, g, width*sizeof(float), streams[g]));
                    // Transfers data device g-1 -> device g
                    cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + dataPerGpu*g+1 , g-1, mat_gpu_tmp[g] + dataPerGpu*g+1, g, width*sizeof(float), streams[g]));
                }
            }
            for(int g = dataLeftover; g < gpus; g++){
                cudaErrorHandle(hipSetDevice(g)); // Unnecessary?
                if(g < gpus-1){
                    // Transfers data device g -> device g+1
                    cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + dataPerGpu*g-width, g+1, mat_gpu_tmp[g] + dataPerGpu*g-width, g, width*sizeof(float), streams[g]));
                    // Transfers data device g-1 -> device g
                    cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + dataPerGpu*g , g-1, mat_gpu_tmp[g] + dataPerGpu*g, g, width*sizeof(float), streams[g]));
                }
                else{
                    // Transfers data device -1 -> device -2
                    cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + dataPerGpu*g, g-1, mat_gpu_tmp[g] + dataPerGpu*g, g, width*sizeof(float), streams[g]));
                }
            }
        }

        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipStreamSynchronize(streams[g]));
        }

        for(int g = 1; g < gpus; g++){
            maxEps_print[0] += maxEps_print[g];
        }

        for(int g = 0; g < gpus; g++){
            float *mat_change = mat_gpu[g];
            mat_gpu[g] = mat_gpu_tmp[g];
            mat_gpu_tmp[g] = mat_change;
        }

        iter--;

        
    }


    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMemcpyAsync(mat + (total/gpus)*g, mat_gpu[g] + (total/gpus)*g, (total/gpus)*sizeof(float), hipMemcpyDeviceToHost, streams[g]));
    }

    end = clock();

    

    cudaErrorHandle(hipDeviceSynchronize());
    
    printf("\nout %i\n", maxEps_print[0]);

    if(iter != 0){
        printf("The computation found a solution with %i gpus. It computed it within %i iterations (%i - %i) and %.3f seconds.\nWidth = %i, Height = %i\nthreadBlock = (%d, %d, %d), gridDim = (%d, %d, %d)\n\n", 
        gpus, print_iter - iter, print_iter, iter, ((double) (end - start)) / CLOCKS_PER_SEC, width, height, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
    }
    else{
        printf("The computation did not find a solution with %i gpus after all its iterations, it ran = %i iterations (%i - %i). It completed it in %.3f seconds.\nWidth = %i, Height = %i\nthreadBlock = (%d, %d, %d), gridDim = (%d, %d, %d)\n\n", 
        gpus, print_iter - iter, print_iter, iter, ((double) (end - start)) / CLOCKS_PER_SEC, width, height, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
    }

    for(int i = 0; i < 20; i++){
        for(int j = 0; j < 20; j++){
            printf("%.3f, ", mat[j + i*width]);
        }
        printf("\n");
    }

    cudaErrorHandle(hipHostFree(mat));
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipFree(mat_gpu[g]));
        cudaErrorHandle(hipFree(mat_gpu_tmp[g]));
        cudaErrorHandle(hipFree(maxEps[g]));
    }
    cudaErrorHandle(hipHostFree(mat_gpu));
    cudaErrorHandle(hipHostFree(mat_gpu_tmp));
    cudaErrorHandle(hipHostFree(maxEps));
    cudaErrorHandle(hipHostFree(device_nr));
    cudaErrorHandle(hipHostFree(maxEps_print));
    cudaErrorHandle(hipHostFree(kernelColl));
}



int main() {
    /*
    Functions   | Type           | Input
    start       | void           | int width, int height, int iter, float eps,
                                   float dx, float dy, dim3 blockDim,
                                   dim3 gridDim

    fillValues  | void           | float *mat, float dx, float dy, int width,
                                   int height

    jacobi      |__global__ void | float *mat_gpu, float *mat_tmp, float eps,
                                   int width, int height, int iter

    ____________________________________________________________________________
    Variables   | Type  | Description
    width       | int   | The width of the matrix
    height      | int   | The height of the matrix
    iter        | int   | Number of max iterations for the jacobian algorithm

    eps         | float | The limit for accepting the state of the matrix during jacobian algorithm
    dx          | float | Distance between each element in the matrix in x direction
    dy          | float | Distance between each element in the matrix in y direction

    blockDim    | dim3  | Number of threads in 3 directions for each block
    gridDim     | dim3  | Number of blocks in 3 directions for the whole grid
    */
    int width = 512;
    int height = 512;
    int iter = 500000;

    float eps = 1.0e-14;
    float dx = 2.0 / (width - 1);
    float dy = 2.0 / (height - 1);

    dim3 blockDim(32, 32, 1);
    dim3 gridDim(16, 1, 1);

    start(width, height, iter, eps, dx, dy, blockDim, gridDim);

    return 0;
}
