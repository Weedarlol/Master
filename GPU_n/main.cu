#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>

#include "jacobi.h"
#include <nvtx3/nvToolsExt.h>

// https://ori-cohen.medium.com/real-life-cuda-programming-part-4-error-checking-e66dcbad6b55
#define cudaErrorHandle(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) 
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort)
            exit(code);
    }
}

void fillValues(double *mat, double dx, double dy, int width, int height){
    double x, y;

    memset(mat, 0, height*width*sizeof(double));

    for(int i = 1; i < height - 1; i++) {
        y = i * dy; // y coordinate
        for(int j = 1; j < width - 1; j++) {
            x = j * dx; // x coordinate
            mat[j + i*width] = sin(M_PI*y)*sin(M_PI*x);
        }
    }
}

void start(int width, int height, int *iter, double eps, double dx, double dy, int gpu_nr, int overlap, int count, dim3 blockDim, dim3 gridDim){
    /*
    Variables   | Type  | Description
    total          | int    | The total number of elements within the matrix

    print_iter     | int    | Placeholder, used to find the number of iterations which has been run

    start          | clock_t| The clock when we reach a certain part of the code
    end            | clock_t| The clock when we are finished with that part of the code

    gpus           | int    | Number of GPUs used when running the program
    device_nr      |*int    | An array which will allow the GPU to see which GPU number it has

    maxEps         |**double| Used by the GPU to calculate how many elements which exceed the allowed limit of change for an element between two iterations
    maxEps_print   |*double | Used as a destination for the number of elements which the GPU has calculated exceeding the limit

    rows_total     | int    | Is the total number of rows which will be computed on by the devices
    rows_per_device| int    | The number of rows per 
    rows_leftover  | int    | The number of GPUs which is required to have an additional row when rows_total cannot be split equally between the devices
    rows_device    | int    | Number of rows each gpus will recieve, including ghostrows
    rows_index     | int    | Contains the index of the row which will be the first row to be transferred to the designated GPU
    rows_compute   | int    | Will tell the GPU how many rows it will compute on

    threadSize     | int    | Total number of threads in work in a GPU
    threadInformatin| int   | Is used to give the GPU information of how many elements each thread has to compute on

    mat            |*double | The matrix which is to be used as base for computations
    mat_gpu        |**double| The matrix which will be a part of the mat matrix, will be the part which is given to one GPU
    mat_gpu_tmp    |**double| Is used so that the GPU can change between the two matrixes every iteration

    streams        |hipStream_t| Contains the streams each GPU will use to allow for asynchronous computing

    kernelColl     |***void | Is a collection of all the functions each GPU will use to run the CUDA function on in a kernel

    filename       | char   | The name of the textdocument which will be created to compare resulting matrixes
    fptr           |*FILE   | Used to create the file
    */


    int total = width*height;

    clock_t start, end;

    int gpus;
    int *device_nr;
    cudaErrorHandle(hipGetDeviceCount(&gpus));
    cudaErrorHandle(hipHostMalloc(&device_nr, gpus*sizeof(int*)));
    for(int g = 0; g < gpus; g++){
        device_nr[g] = g;
    }

    int **maxEps, *maxEps_print;
    cudaErrorHandle(hipHostMalloc(&maxEps,       gpus*sizeof(int*)));
    cudaErrorHandle(hipHostMalloc(&maxEps_print, gpus*sizeof(int)));


    // Ignores first and last row
    int rows_total = height-2;
    int rows_per_device = rows_total/gpus;
    int rows_leftover = rows_total%gpus;
    int *rows_device, *rows_index, *rows_compute;
    cudaErrorHandle(hipHostMalloc(&rows_device, gpus*sizeof(int*)));
    cudaErrorHandle(hipHostMalloc(&rows_index, gpus*sizeof(int*)));
    cudaErrorHandle(hipHostMalloc(&rows_compute, gpus*sizeof(int*)));
    // Calculate the number of rows for each device
    for (int g = 0; g < gpus; g++) {
        int extra_row = (g < rows_leftover) ? 1 : 0;
  
        rows_device[g] = rows_per_device + extra_row + 2;

        rows_compute[g] = rows_per_device + extra_row;

        rows_index[g] = g * rows_per_device + min(g, rows_leftover);
    }

    int *threadInformation;
    int threadSize = blockDim.x*blockDim.y*blockDim.z*gridDim.x*gridDim.y*gridDim.z;
    cudaErrorHandle(hipHostMalloc(&threadInformation, 4*sizeof(int)));
    threadInformation[0] = (rows_compute[0]     *(width-2))/threadSize; // Find number of elements to compute for each thread, ignoring border elements.
    threadInformation[1] = (rows_compute[0]     *(width-2))%threadSize; // Finding which threads require 1 more element
    threadInformation[2] = (rows_compute[gpus-1]*(width-2))/threadSize; // Find number of elements to compute for each thread, ignoring border elements. -1 because of ghost row
    threadInformation[3] = (rows_compute[gpus-1]*(width-2))%threadSize; // Finding which threads require 1 more element

    double *mat;
    double **mat_gpu, **mat_gpu_tmp;
    cudaErrorHandle(hipHostMalloc(&mat,          total*sizeof(double)));
    cudaErrorHandle(hipHostMalloc(&mat_gpu,      gpus*sizeof(double*)));
    cudaErrorHandle(hipHostMalloc(&mat_gpu_tmp,  gpus*sizeof(double*)));
    // Allocates memory on devices based on number of rows for each device
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMalloc(&mat_gpu[g],     width*rows_device[g]*sizeof(double)));
        cudaErrorHandle(hipMalloc(&mat_gpu_tmp[g], width*rows_device[g]*sizeof(double)));
        cudaErrorHandle(hipMalloc(&maxEps[g],      threadSize*sizeof(int*)));
        maxEps_print[g] = 1;
    }


    hipStream_t streams[gpus];
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipStreamCreate(&streams[g]));
    }
    

    fillValues(mat, dx, dy, width, height);


    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMemcpyAsync(mat_gpu[g], mat+(rows_index[g])*width, rows_device[g]*width*sizeof(double), hipMemcpyHostToDevice, streams[g]));
    }

    void ***kernelColl;
    cudaErrorHandle(hipHostMalloc(&kernelColl, gpus * sizeof(void**)));
    // Allocates the elements in the kernelColl, used for hipLaunchCooperativeKernel as functon variables.
    for (int g = 0; g < gpus; g++) {
        void **kernelArgs = new void*[13];
        kernelArgs[0] = &mat_gpu[g];
        kernelArgs[1] = &mat_gpu_tmp[g];
        kernelArgs[2] = &rows_device[g]; // How many rows for each device
        kernelArgs[3] = &width;
        kernelArgs[4] = &height;
        kernelArgs[5] = &rows_leftover; // Tells how many of the devices will have 1 extra row
        kernelArgs[6] = &device_nr[g];
        kernelArgs[7] = &rows_compute[g];
        kernelArgs[8] = &threadInformation[0];
        kernelArgs[9] = &threadInformation[1];
        kernelArgs[10] = &threadInformation[2];
        kernelArgs[11] = &threadInformation[3];
        kernelArgs[12] = &maxEps[g];
        kernelArgs[13] = &eps;

        kernelColl[g] = kernelArgs;
    }



    // FØR __________________________________________________________
    for(int i = count-1; i > 0; i--){
        iter[i] -= iter[i-1];
    }


    start = clock();
    if(overlap == 1){
        for(int i = 0; i < count; i++){
            int it = iter[i];
            while(it > 0 && maxEps_print[0] != 0){
                for(int g = 0; g < gpus; g++){
                    cudaErrorHandle(hipSetDevice(g));
                    cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobi, gridDim, blockDim, kernelColl[g], 0, streams[g]));
                    cudaErrorHandle(hipMemcpyAsync(&maxEps_print[g], maxEps[g], sizeof(int), hipMemcpyDeviceToHost, streams[g]));
                }
                for(int g = 0; g < gpus; g++){
                    cudaErrorHandle(hipStreamSynchronize(streams[g]));
                }
                if(gpus > 1){
                    for(int g = 0; g < gpus; g++){
                        cudaErrorHandle(hipSetDevice(g));
                        if(g == 0){
                            // Transfers data device 0 -> device 1
                            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[1],                                   1, mat_gpu_tmp[0] + (rows_compute[0])*width, 0, width*sizeof(double), streams[g]));
                        }
                        else if(g < gpus-1){
                            // Transfers data device g -> device g+1
                            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1],                               g+1, mat_gpu_tmp[g] + (rows_compute[g])*width, g, width*sizeof(double), streams[g]));
                            // Transfers data device g -> device g-1
                            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (rows_compute[g-1]+1)*width, g-1, mat_gpu_tmp[g] + width,                   g, width*sizeof(double), streams[g]));
                        }
                        else{
                            // Transfers data device -1 -> device -2
                            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (rows_compute[g-1]+1)*width, g-1, mat_gpu_tmp[g] + width,                   g, width*sizeof(double), streams[g]));
                        }  
                    }
                }
                for(int g = 0; g < gpus; g++){
                    cudaErrorHandle(hipStreamSynchronize(streams[g]));
                }
                for(int g = 1; g < gpus; g++){
                    maxEps_print[0] += maxEps_print[g];
                }
                for(int g = 0; g < gpus; g++){
                    double *mat_change = mat_gpu[g];
                    mat_gpu[g] = mat_gpu_tmp[g];
                    mat_gpu_tmp[g] = mat_change;
                }
                it--;
            }
            iter[i] = (i == 0) ? iter[0] : iter[i] + iter[i-1];
            printf("%.6f, %i, %i, %i\n", ((double) (clock() - start)) / CLOCKS_PER_SEC, iter[i], iter[i] - it, (it == 0) ? 0 : 1);
        }
    }
    if(overlap == 1){
        for(int i = 0; i < count; i++){
            int it = iter[i];
            while(it > 0 && maxEps_print[0] != 0){
                for(int g = 0; g < gpus; g++){
                    cudaErrorHandle(hipSetDevice(g));
                    cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobi, gridDim, blockDim, kernelColl[g], 0, streams[g]));
                    cudaErrorHandle(hipMemcpyAsync(&maxEps_print[g], maxEps[g], sizeof(int), hipMemcpyDeviceToHost, streams[g]));
                }
                for(int g = 0; g < gpus; g++){
                    cudaErrorHandle(hipStreamSynchronize(streams[g]));
                }
                if(gpus > 1){
                    for(int g = 0; g < gpus; g++){
                        cudaErrorHandle(hipSetDevice(g));
                        if(g == 0){
                            // Transfers data device 0 -> device 1
                            cudaErrorHandle(hipMemcpyPeer(mat_gpu_tmp[1],                                   1, mat_gpu_tmp[0] + (rows_compute[0])*width, 0, width*sizeof(double)));
                        }
                        else if(g < gpus-1){
                            // Transfers data device g -> device g+1
                            cudaErrorHandle(hipMemcpyPeer(mat_gpu_tmp[g+1],                               g+1, mat_gpu_tmp[g] + (rows_compute[g])*width, g, width*sizeof(double)));
                            // Transfers data device g -> device g-1
                            cudaErrorHandle(hipMemcpyPeer(mat_gpu_tmp[g-1] + (rows_compute[g-1]+1)*width, g-1, mat_gpu_tmp[g] + width,                   g, width*sizeof(double)));
                        }
                        else{
                            // Transfers data device -1 -> device -2
                            cudaErrorHandle(hipMemcpyPeer(mat_gpu_tmp[g-1] + (rows_compute[g-1]+1)*width, g-1, mat_gpu_tmp[g] + width,                   g, width*sizeof(double)));
                        }  
                    }
                }
                for(int g = 0; g < gpus; g++){
                    cudaErrorHandle(hipStreamSynchronize(streams[g]));
                }
                for(int g = 1; g < gpus; g++){
                    maxEps_print[0] += maxEps_print[g];
                }
                for(int g = 0; g < gpus; g++){
                    double *mat_change = mat_gpu[g];
                    mat_gpu[g] = mat_gpu_tmp[g];
                    mat_gpu_tmp[g] = mat_change;
                }
                it--;
            }
            iter[i] = (i == 0) ? iter[0] : iter[i] + iter[i-1];
            printf("%.6f, %i, %i, %i, %i\n", ((double) (clock() - start)) / CLOCKS_PER_SEC, iter[i], (it == 0) ? 0 : 1, iter[i] - it, overlap);
        }
    }


    




    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipStreamSynchronize(streams[g]));
    }
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipMemcpyAsync(mat + (rows_index[g]+1)*width, mat_gpu[g] + width, rows_compute[g]*width*sizeof(double), hipMemcpyDeviceToHost, streams[g]));
    }
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipStreamSynchronize(streams[g]));
    }


    /* if(iter != 0){
        printf("The computation found a solution with %i gpus. It computed it within %i iterations (%i - %i) and %.3f seconds.\nWidth = %i, Height = %i\nthreadBlock = (%d, %d, %d), gridDim = (%d, %d, %d)\n\n", 
        gpus, print_iter - iter, print_iter, iter, ((double) (end - start)) / CLOCKS_PER_SEC, width, height, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
    }
    else{
        printf("The computation did not find a solution with %i gpus after all its iterations, it ran = %i iterations (%i - %i). It completed it in %.3f seconds.\nWidth = %i, Height = %i\nthreadBlock = (%d, %d, %d), gridDim = (%d, %d, %d)\n\n", 
        gpus, print_iter - iter, print_iter, iter, ((double) (end - start)) / CLOCKS_PER_SEC, width, height, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
    }

    printf("etter\n%i\n", gpu_nr);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipStreamSynchronize(streams[g]));
    }

    for(int g = 0; g < gpus; g++){
        printf("%i -> index = %i, device = %i, compute = %i, leftover = %i, element_per_thread = %i, extra_element = %i\n", 
        g, rows_index[g], rows_device[g], rows_compute[g], rows_leftover, threadInformation[2], threadInformation[3]);
    } */




    // Creates an output which can be used to compare the different resulting matrixes
    /* FILE *fptr;
    char filename[30];
    sprintf(filename, "mat/GPU_%i_Matrix%i_%i.txt", gpu_nr, width, height);
    fptr = fopen(filename, "w");
    for(int i = 0; i < height; i++){
        for(int j = 0; j < width; j++){
            fprintf(fptr, "%.16f ", mat[j + i*width]);
        }
        fprintf(fptr, "\n");
    }
    fclose(fptr); */


    // Prints to use for creating 
    /* printf("Time - %.4f, SolutionFound - %s, IterationsComputed - %i",
            ((double) (end - start)) / CLOCKS_PER_SEC, (iter[0] == 0) ? "No" : "Yes", iter[0]);  */




    // Frees up memory as we are finished with the program
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipFree(mat_gpu[g]));
        cudaErrorHandle(hipFree(mat_gpu_tmp[g]));
        cudaErrorHandle(hipFree(maxEps[g]));
        cudaErrorHandle(hipStreamDestroy(streams[g]));
    }
    cudaErrorHandle(hipHostFree(mat));
    cudaErrorHandle(hipHostFree(mat_gpu));
    cudaErrorHandle(hipHostFree(mat_gpu_tmp));
    cudaErrorHandle(hipHostFree(threadInformation));
    cudaErrorHandle(hipHostFree(device_nr));
    cudaErrorHandle(hipHostFree(maxEps));
    cudaErrorHandle(hipHostFree(maxEps_print));
    cudaErrorHandle(hipHostFree(rows_device));
    cudaErrorHandle(hipHostFree(rows_index));
    cudaErrorHandle(hipHostFree(rows_compute));
}



int main(int argc, char *argv[]) {
    /*
    Functions   | Type           | Input
    start       | void           | int width, int height, int iter, double eps,
                                   double dx, double dy, dim3 blockDim,
                                   dim3 gridDim

    fillValues  | void           | double *mat, double dx, double dy, int width,
                                   int height

    jacobi      |__global__ void | double *mat_gpu, double *mat_tmp, double eps,
                                   int width, int height, int iter

    ____________________________________________________________________________
    Variables   | Type  | Description
    width       | int   | The width of the matrix
    height      | int   | The height of the matrix
    iter        | int   | Number of max iterations for the jacobian algorithm

    eps         | double | The limit for accepting the state of the matrix during jacobian algorithm
    dx          | double | Distance between each element in the matrix in x direction
    dy          | double | Distance between each element in the matrix in y direction

    blockDim    | dim3  | Number of threads in 3 directions for each block
    gridDim     | dim3  | Number of blocks in 3 directions for the whole grid
    */
    if (argc != 6) {
        printf("Usage: %s <Width> <Height> <Iterations>", argv[0]); // Programname
        return 1;
    }

    int width = atoi(argv[1]);
    int height = atoi(argv[2]);
    int gpu_nr = atoi(argv[4]);
    int overlap = atoi(argv[5]);

    // When working on several iterations
    int *iter;
    hipHostMalloc(&iter, 10*sizeof(int));
    char *token;
    token = strtok(argv[3], "_");
    int count = 0;

    while(token != NULL && count < 10){
        iter[count] = atoi(token);
        count++;
        token = strtok(NULL, "_");
    }



    double eps = 1.0e-14;
    double dx = 2.0 / (width - 1);
    double dy = 2.0 / (height - 1);

    dim3 blockDim(32, 32, 1);
    dim3 gridDim(16, 1, 1);

    start(width, height, iter, eps, dx, dy, gpu_nr, overlap, count, blockDim, gridDim);

    return 0;
}
