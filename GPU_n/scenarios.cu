#include <stdio.h>
#include <math.h>


#include "errorHandle.h"
#include "jacobi.h"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

void initializeStreamsAndEvents(int gpus, hipStream_t streams[][2], hipEvent_t events[][4], hipEvent_t *startevent, hipEvent_t *stopevent){
    for (int g = 0; g < gpus; g++) {
        hipSetDevice(g);
        cudaErrorHandle(hipStreamCreate(&streams[g][0]));
        cudaErrorHandle(hipStreamCreate(&streams[g][1]));
        cudaErrorHandle(hipEventCreate(&events[g][0]));
        cudaErrorHandle(hipEventCreate(&events[g][1]));
        cudaErrorHandle(hipEventCreate(&events[g][2]));
        cudaErrorHandle(hipEventCreate(&events[g][3]));
    }
    cudaErrorHandle(hipEventCreate(startevent));
    cudaErrorHandle(hipEventCreate(stopevent));
}

void freeStreamsAndEvents(int gpus, hipStream_t streams[][2], hipEvent_t events[][4], hipEvent_t *startevent, hipEvent_t *stopevent) {
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipStreamDestroy(streams[g][0]));
        cudaErrorHandle(hipStreamDestroy(streams[g][1]));
        cudaErrorHandle(hipEventDestroy(events[g][0]));
        cudaErrorHandle(hipEventDestroy(events[g][1]));
        cudaErrorHandle(hipEventDestroy(events[g][2]));
        cudaErrorHandle(hipEventDestroy(events[g][3]));
    }
}





void full_calculation_overlap(double **mat_gpu, double **mat_gpu_tmp, int height, int width, int iter, int gpus, int *rows_device, dim3 gridDim, dim3 blockDim, void*** kernelCollEdge, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 1
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            // Computes the upper and lower row
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiEdge, gridDim, blockDim, kernelCollEdge[g], 0, streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
            // Computes the rest of the rows
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
        }


        // Step 2
        // Transfer 2 row of the matrix
        for(int g = 1; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipStreamWaitEvent(streams[g][0], events[g][0]));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (rows_device[g-1]-1)*width + 1, g-1, mat_gpu_tmp[g] + width + 1, g, (width-2)*sizeof(double), streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][2], streams[g][0]));
        }
        // Transfers n-2 row of the matrix
        for(int g = 0; g < gpus-1; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipStreamWaitEvent(streams[g][0], events[g][0]));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + 1, g+1, mat_gpu_tmp[g] + (rows_device[g]-2)*width + 1, g, (width-2)*sizeof(double), streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][3], streams[g][0]));
        }


        // Step 3
        for (int g = 0; g < gpus; g++) {
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
            cudaErrorHandle(hipEventSynchronize(events[g][2]));
            cudaErrorHandle(hipEventSynchronize(events[g][3]));
        }
        
        // Step 4
        for(int g = 0; g < gpus; g++){
            double *mat_change = mat_gpu[g];
            mat_gpu[g] = mat_gpu_tmp[g];
            mat_gpu_tmp[g] = mat_change;
        }
        iter--;
    }

    cudaErrorHandle(hipEventRecord(stopevent));
    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void full_calculation_nooverlap(double **mat_gpu, double **mat_gpu_tmp, int height, int width, int iter, int gpus, int *rows_device, dim3 gridDim, dim3 blockDim, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 1
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }

        // Step 2
        // Transfers 2 row of the matrix
        for(int g = 1; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (rows_device[g-1]-1)*width + 1, g-1, mat_gpu_tmp[g] + width + 1, g, (width-2)*sizeof(double), streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
        }
        // Transfers n-2 row of the matrix
        for(int g = 0; g < gpus-1; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + 1, g+1, mat_gpu_tmp[g] + (rows_device[g]-2)*width + 1, g, (width-2)*sizeof(double), streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][2], streams[g][1]));
        }

        // Step 3
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
            cudaErrorHandle(hipEventSynchronize(events[g][2]));
        }
        
        // Step 4
        for(int g = 0; g < gpus; g++){
            double *mat_change = mat_gpu[g];
            mat_gpu[g] = mat_gpu_tmp[g];
            mat_gpu_tmp[g] = mat_change;
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void no_kernel_overlap(double **mat_gpu, double **mat_gpu_tmp, int height, int width, int iter, int gpus, int *rows_device, dim3 gridDim, dim3 blockDim, void*** kernelCollEdge, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Transfer 2 row of the matrix
        for(int g = 1; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (rows_device[g-1]-1)*width + 1, g-1, mat_gpu_tmp[g] + width + 1, g, (width-2)*sizeof(double), streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }
        // Transfers n-2 row of the matrix
        for(int g = 0; g < gpus-1; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + 1, g+1, mat_gpu_tmp[g] + (rows_device[g]-2)*width + 1, g, (width-2)*sizeof(double), streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
        }

        // Step 3
        for (int g = 0; g < gpus; g++) {
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
        }
        
        // Step 4
        for(int g = 0; g < gpus; g++){
            double *mat_change = mat_gpu[g];
            mat_gpu[g] = mat_gpu_tmp[g];
            mat_gpu_tmp[g] = mat_change;
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));
    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void no_kernel_nooverlap(double **mat_gpu, double **mat_gpu_tmp, int height, int width, int iter, int gpus, int *rows_device, dim3 gridDim, dim3 blockDim, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 2
        // Transfers 2 row of the matrix
        for(int g = 1; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (rows_device[g-1]-1)*width + 1, g-1, mat_gpu_tmp[g] + width + 1, g, (width-2)*sizeof(double), streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }
        // Transfers n-2 row of the matrix
        for(int g = 0; g < gpus-1; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + 1, g+1, mat_gpu_tmp[g] + (rows_device[g]-2)*width + 1, g, (width-2)*sizeof(double), streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
        }

        // Step 3
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
        }
        
        // Step 4
        for(int g = 0; g < gpus; g++){
            double *mat_change = mat_gpu[g];
            mat_gpu[g] = mat_gpu_tmp[g];
            mat_gpu_tmp[g] = mat_change;
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void no_communication_overlap(double **mat_gpu, double **mat_gpu_tmp, int height, int width, int iter, int gpus, int *rows_device, dim3 gridDim, dim3 blockDim, void*** kernelCollEdge, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 1
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiEdge, gridDim, blockDim, kernelCollEdge[g], 0, streams[g][0]));
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][0]));
        }

        for (int g = 0; g < gpus; g++) {
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
        }
        
        for(int g = 0; g < gpus; g++){
            double *mat_change = mat_gpu[g];
            mat_gpu[g] = mat_gpu_tmp[g];
            mat_gpu_tmp[g] = mat_change;
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));
    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void no_communication_nooverlap(double **mat_gpu, double **mat_gpu_tmp, int height, int width, int iter, int gpus, int *rows_device, dim3 gridDim, dim3 blockDim, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 1
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }

        // Step 3
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
        }
        
        // Step 4
        for(int g = 0; g < gpus; g++){
            double *mat_change = mat_gpu[g];
            mat_gpu[g] = mat_gpu_tmp[g];
            mat_gpu_tmp[g] = mat_change;
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void only_events(double **mat_gpu, double **mat_gpu_tmp, int height, int width, int iter, int gpus, int *rows_device, dim3 gridDim, dim3 blockDim, void*** kernelCollMid){

    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
         for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }

        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
        }

        for(int g = 0; g < gpus; g++){
            double *mat_change = mat_gpu[g];
            mat_gpu[g] = mat_gpu_tmp[g];
            mat_gpu_tmp[g] = mat_change;
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void only_calculation_overlap(double **mat_gpu, double **mat_gpu_tmp, int height, int width, int iter, int gpus, int *rows_device, dim3 gridDim, dim3 blockDim, void*** kernelCollEdge, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 1
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            // Computes the upper and lower row
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiEdge, gridDim, blockDim, kernelCollEdge[g], 0, streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
            // Computes the rest of the rows
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
        }

        // Step 3
        for (int g = 0; g < gpus; g++) {
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
        }
        
        iter--;
    }
    

    cudaErrorHandle(hipEventRecord(stopevent));
    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void only_calculation_nooverlap(double **mat_gpu, double **mat_gpu_tmp, int height, int width, int iter, int gpus, int *rows_device, dim3 gridDim, dim3 blockDim, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 1
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }

        // Step 3
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
        }

        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void only_communication_overlap(double **mat_gpu, double **mat_gpu_tmp, int height, int width, int iter, int gpus, int *rows_device, dim3 gridDim, dim3 blockDim, void*** kernelCollEdge, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Transfer 2 row of the matrix
        for(int g = 1; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (rows_device[g-1]-1)*width + 1, g-1, mat_gpu_tmp[g] + width + 1, g, (width-2)*sizeof(double), streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }
        // Transfers n-2 row of the matrix
        for(int g = 0; g < gpus-1; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + 1, g+1, mat_gpu_tmp[g] + (rows_device[g]-2)*width + 1, g, (width-2)*sizeof(double), streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
        }

        // Step 3
        for (int g = 0; g < gpus; g++) {
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));
    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void only_communication_nooverlap(double **mat_gpu, double **mat_gpu_tmp, int height, int width, int iter, int gpus, int *rows_device, dim3 gridDim, dim3 blockDim, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 2
        // Transfers 2 row of the matrix
        for(int g = 1; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (rows_device[g-1]-1)*width + 1, g-1, mat_gpu_tmp[g] + width + 1, g, (width-2)*sizeof(double), streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }
        // Transfers n-2 row of the matrix
        for(int g = 0; g < gpus-1; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + 1, g+1, mat_gpu_tmp[g] + (rows_device[g]-2)*width + 1, g, (width-2)*sizeof(double), streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
        }

        // Step 3
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}

