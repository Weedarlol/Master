#include "hip/hip_runtime.h"
// cuda_utils.cu

#include "global_functions.h"

void fillValues(double *mat, double dx, double dy, int width, int height) {
    double x, y;

    memset(mat, 0, height * width * sizeof(double));

    for (int i = 1; i < height - 1; i++) {
        y = i * dy; // y coordinate
        for (int j = 1; j < width - 1; j++) {
            x = j * dx; // x coordinate
            mat[j + i * width] = sin(M_PI * y) * sin(M_PI * x);
        }
    }
}

void fillValues3D(double *mat, int width, int height, int depth_node, double dx, double dy, double dz, int rank) {
    double x, y, z;

    // Assuming the data in the matrix is stored contiguously in memory
    memset(mat, 0, height * width * depth_node * sizeof(double));

    for (int i = 1; i < depth_node-1; i++) {
        z = i * dz; // z coordinate
        for (int j = 1; j < height - 1; j++) {
            y = j * dy; // z coordinate
            for (int k = 1; k < width - 1; k++) {
                x = k * dx; // x coordinate
                mat[k +  j*width + i*width*height] = sin(M_PI * x) * sin(M_PI * y) * sin(M_PI * z);
            }
        }
    }
}

void initializeStreamsAndEvents(int gpus, hipStream_t streams[][2], hipEvent_t events[][4], hipEvent_t *startevent, hipEvent_t *stopevent) {
    for (int g = 0; g < gpus; g++) {
        hipSetDevice(g);
        cudaErrorHandle(hipStreamCreate(&streams[g][0]));
        cudaErrorHandle(hipStreamCreate(&streams[g][1]));
        cudaErrorHandle(hipEventCreate(&events[g][0]));
        cudaErrorHandle(hipEventCreate(&events[g][1]));
        cudaErrorHandle(hipEventCreate(&events[g][2]));
        cudaErrorHandle(hipEventCreate(&events[g][3]));
    }
    cudaErrorHandle(hipEventCreate(startevent));
    cudaErrorHandle(hipEventCreate(stopevent));
}

void freeStreamsAndEvents(int gpus, hipStream_t streams[][2], hipEvent_t events[][4], hipEvent_t *startevent, hipEvent_t *stopevent) {
    for (int g = 0; g < gpus; g++) {
        cudaErrorHandle(hipStreamDestroy(streams[g][0]));
        cudaErrorHandle(hipStreamDestroy(streams[g][1]));
        cudaErrorHandle(hipEventDestroy(events[g][0]));
        cudaErrorHandle(hipEventDestroy(events[g][1]));
        cudaErrorHandle(hipEventDestroy(events[g][2]));
        cudaErrorHandle(hipEventDestroy(events[g][3]));
    }
}
