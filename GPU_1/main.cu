#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>

#include "jacobi.h"
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;
namespace cg = cooperative_groups;


// https://ori-cohen.medium.com/real-life-cuda-programming-part-4-error-checking-e66dcbad6b55
#define cudaErrorHandle(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) 
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort)
            exit(code);
    }
}

void fillValues(double *mat, double dx, double dy, int width, int height){
    double x, y;

    memset(mat, 0, height*width*sizeof(double));

    for(int i = 1; i < height - 1; i++) {
        y = i * dy; // y coordinate
        for(int j = 1; j < width - 1; j++) {
            x = j * dx; // x coordinate
            mat[j + i*width] = sin(M_PI*y)*sin(M_PI*x);
        }
    }
}






void start(int width, int height, int iter, double eps, double dx, double dy, dim3 blockDim, dim3 gridDim){
    /*
    Variables   | Type  | Description
    total       | int   | Total number of elements in the matrix
    dataPerGpu  | int   | Number of elements per available gpu

    start       |clock_t| Start timer of program
    end         |clock_t| End timer of program

    mat         |*double | Pointer to the allocated matrix in the CPU
    mat_gpu     |**double| Pointer to an allocated matrix in the GPU
    mat_gpu_tmp |**double| Pointer to an allocated matrix in the GPU
    maxEps      |*int   | Pointer to an allocated vector in the GPU used for checking if the matrix is in an acceptable state
    comp_suc    |*int   | Checks if the computation is successfull or not
    */

    int total = width*height;
    int print_iter = iter;
    clock_t start, end;


    double *mat, *mat_gpu, *mat_gpu_tmp;
    cudaErrorHandle(hipHostMalloc(&mat, total*sizeof(double)));
    cudaErrorHandle(hipMalloc(&mat_gpu, total*sizeof(double*)));
    cudaErrorHandle(hipMalloc(&mat_gpu_tmp, total*sizeof(double*)));
    

    int *maxEps, *comp_suc;;
    cudaErrorHandle(hipMalloc(&maxEps, blockDim.x*blockDim.y*blockDim.z*gridDim.x*gridDim.y*gridDim.z*sizeof(int)));
    cudaErrorHandle(hipHostMalloc(&comp_suc, sizeof(int*)));


    /* initialization */
    fillValues(mat, dx, dy, width, height);
    





    // Here we are done with the allocation, and start with the compution
    start = clock();

    // Copies elemts over from CPU to the device.
    cudaErrorHandle(hipMemcpyAsync(mat_gpu, mat, total*sizeof(double), hipMemcpyHostToDevice));
    cudaErrorHandle(hipMemsetAsync(mat_gpu_tmp, 0, total*sizeof(double)));

    // Creates an array where its elements are features in hipLaunchCooperativeKernel
    void *kernelArgs[] = {&mat_gpu, &mat_gpu_tmp, &eps, &width, &height, &iter, &maxEps};


    // Runs device
    // jacobi<<<gridDim, blockDim>>>(mat_gpu, mat_gpu_tmp, eps, width, height, iter, maxEps);
    cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobi, gridDim, blockDim, kernelArgs));

    cudaErrorHandle(hipDeviceSynchronize());

    // Copies back value from device i to CPU
    cudaErrorHandle(hipMemcpy(mat, mat_gpu, total*sizeof(double), hipMemcpyDeviceToHost));
    
    cudaErrorHandle(hipMemcpy(comp_suc, maxEps, sizeof(int*), hipMemcpyDeviceToHost));

    cudaErrorHandle(hipDeviceSynchronize());

    end = clock();







    if(*comp_suc != 0){
        printf("The computation found a solution. It computed it within %i iterations (%i - %i) and %.3f seconds.\nWidth = %i, Height = %i\nthreadBlock = (%d, %d, %d), gridDim = (%d, %d, %d)\n\n", 
        print_iter - *comp_suc, print_iter, *comp_suc, ((double) (end - start)) / CLOCKS_PER_SEC, width, height, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
    }
    else{
        printf("The computation did not find a solution after all its iterations, it ran = %i iterations (%i - %i). It completed it in %.3f seconds.\nWidth = %i, Height = %i\nthreadBlock = (%d, %d, %d), gridDim = (%d, %d, %d)\n\n", 
        print_iter - *comp_suc, print_iter, *comp_suc, ((double) (end - start)) / CLOCKS_PER_SEC, width, height, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
    }

    cudaErrorHandle(hipHostFree(mat));
    cudaErrorHandle(hipFree(mat_gpu));
    cudaErrorHandle(hipFree(mat_gpu_tmp));
}



int main(int argc, char *argv[]) {
    /*
    Functions   | Type           | Input
    start       | void           | int width, int height, int iter, double eps,
                                   double dx, double dy, dim3 blockDim,
                                   dim3 gridDim

    fillValues  | void           | double *mat, double dx, double dy, int width,
                                   int height

    jacobi      |__global__ void | double *mat_gpu, double *mat_tmp, double eps,
                                   int width, int height, int iter

    ____________________________________________________________________________
    Variables   | Type  | Description
    width       | int   | The width of the matrix
    height      | int   | The height of the matrix
    iter        | int   | Number of max iterations for the jacobian algorithm

    eps         | double | The limit for accepting the state of the matrix during jacobian algorithm
    dx          | double | Distance between each element in the matrix in x direction
    dy          | double | Distance between each element in the matrix in y direction

    blockDim    | dim3  | Number of threads in 3 directions for each block
    gridDim     | dim3  | Number of blocks in 3 directions for the whole grid
    */
    if (argc != 4) {
        printf("Usage: %s <Width> <Height> <Iterations>", argv[0]); // Programname
        return 1;
    }

    int width = atoi(argv[1]);
    int height = atoi(argv[2]);
    int iter = atoi(argv[3]);

    double eps = 1.0e-14;
    double dx = 2.0 / (width - 1);
    double dy = 2.0 / (height - 1);

    dim3 blockDim(32, 32, 1);
    dim3 gridDim(16, 1, 1);

    start(width, height, iter, eps, dx, dy, blockDim, gridDim);

    return 0;
}
