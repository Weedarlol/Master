#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// https://ori-cohen.medium.com/real-life-cuda-programming-part-4-error-checking-e66dcbad6b55
#define cudaErrorHandle(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) 
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort)
            exit(code);
    }
}

#define N 1000

int main() {
    int *h_a, *h_b;
    int *d_a, *d_b;

    float milliseconds = 0;

    hipEvent_t start, stop;

    // Allocate memory on the host (CPU)
    cudaErrorHandle(hipHostMalloc(&h_a, N*sizeof(int), hipHostMallocDefault));
    cudaErrorHandle(hipHostMalloc(&h_a, N*sizeof(int), hipHostMallocDefault));

    // Allocate memory on the device (GPU)
    cudaErrorHandle(hipMalloc((void**)&d_a, N * sizeof(int)));
    cudaErrorHandle(hipMalloc((void**)&d_b, N * sizeof(int)));

    // Initialize data on the host
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
    }

    // Record start time
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Transfer data from host to device
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);

    // Transfer data from device to host
    hipMemcpy(h_b, d_b, N * sizeof(int), hipMemcpyDeviceToHost);

    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Latency: %f milliseconds\n", milliseconds);

    // Clean up
    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);

    return 0;
}

