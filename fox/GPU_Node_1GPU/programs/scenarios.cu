#include <stdio.h>
#include <math.h>
#include <mpi.h>

#include "cuda_functions.h"
#include "jacobi.h"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

void full_calculation(double **data_gpu, double **data_gpu_tmp, int width, int height, int depth_node, int iter, int *slices_device, int rank, dim3 gridDim, dim3 blockDim, void** kernelCollMid){
    hipStream_t streams[2];
    hipEvent_t events, startevent, stopevent;
    cudaErrorHandle(hipStreamCreate(&streams[0]));
    cudaErrorHandle(hipStreamCreate(&streams[1]));
    cudaErrorHandle(hipEventCreate(&events));
    cudaErrorHandle(hipEventCreate(&startevent));
    cudaErrorHandle(hipEventCreate(&stopevent));
    MPI_Request myRequest[2];
    MPI_Status myStatus[2];

    cudaErrorHandle(hipDeviceSynchronize());
    cudaErrorHandle(hipEventRecord(startevent));
    

    while(iter > 0){
        cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobi, gridDim, blockDim, kernelCollMid, 0, streams[0]));
        cudaErrorHandle(hipEventRecord(events, streams[0]));
        
        cudaErrorHandle(hipEventSynchronize(events));

        /* if(rank == 0){
            MPI_Isend(&data_gpu_tmp[width*height*(depth_node-2)], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[0]);
            MPI_Irecv(&data_gpu_tmp[width*height*(depth_node-1)], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &myRequest[1]); 

            MPI_Waitall(2, myRequest, myStatus);
        }
        else if(rank == size-1){
            MPI_Irecv(&data_gpu_tmp[0],                           width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[rank*2-2]); 
            MPI_Isend(&data_gpu_tmp[width*height],                width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, &myRequest[rank*2-1]); 

            MPI_Waitall(2, &myRequest[rank*2-2], myStatus);
        }
        MPI_Barrier(MPI_COMM_WORLD); */
            
        double *data_change = data_gpu[0];
        data_gpu[0] = data_gpu_tmp[0];
        data_gpu_tmp[0] = data_change;
        iter--;
    }

    printf("Finished full computation loop\n");

    cudaErrorHandle(hipDeviceSynchronize());

    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    cudaErrorHandle(hipDeviceSynchronize());
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.5f s\n", milliseconds/1000);

    //freeStreamsAndEvents(1, streams, events, &startevent, &stopevent);
}