#include <stdio.h>
#include <math.h>


#include "../../global_functions.h"
#include "jacobi.h"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;




/* 
void full_calculation_overlap(double **mat_gpu, double **mat_gpu_tmp, int width, int height, int depth, int iter, int gpus, int *slices_device, dim3 gridDim, dim3 blockDim, void*** kernelCollEdge, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 1
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            // Computes the upper and lower slice
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiEdge, gridDim, blockDim, kernelCollEdge[g], 0, streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
            // Computes the rest of the slices
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
        }


        // Step 2
        // Transfer 2 slice of the matrix
        for(int g = 1; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipStreamWaitEvent(streams[g][0], events[g][0]));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (slices_device[g-1]-1)*width + 1, g-1, mat_gpu_tmp[g] + width + 1, g, (width-2)*(height-2)*sizeof(double), streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][2], streams[g][0]));
        }
        // Transfers n-2 slice of the matrix
        for(int g = 0; g < gpus-1; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipStreamWaitEvent(streams[g][0], events[g][0]));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + 1, g+1, mat_gpu_tmp[g] + (slices_device[g]-2)*width + 1, g, (width-2)*(height-2)*sizeof(double), streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][3], streams[g][0]));
        }


        // Step 3
        for (int g = 0; g < gpus; g++) {
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
            cudaErrorHandle(hipEventSynchronize(events[g][2]));
            cudaErrorHandle(hipEventSynchronize(events[g][3]));
        }
        
        // Step 4
        for(int g = 0; g < gpus; g++){
            double *mat_change = mat_gpu[g];
            mat_gpu[g] = mat_gpu_tmp[g];
            mat_gpu_tmp[g] = mat_change;
        }
        iter--;
    }

    cudaErrorHandle(hipEventRecord(stopevent));
    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}
 */


void full_calculation_nooverlap(double **mat_gpu, double **mat_gpu_tmp, int width, int height, int depth, int iter, int gpus, int *slices_device, dim3 gridDim, dim3 blockDim, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 1
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }

        // Step 2
        // Transfers 2 slice of the matrix
        for(int g = 1; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (slices_device[g-1]-1)*width + 1, g-1, mat_gpu_tmp[g] + width + 1, g, (width-2)*sizeof(double), streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
        }
        // Transfers n-2 slice of the matrix
        for(int g = 0; g < gpus-1; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipStreamWaitEvent(streams[g][1], events[g][0]));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + 1, g+1, mat_gpu_tmp[g] + (slices_device[g]-2)*width + 1, g, (width-2)*sizeof(double), streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][2], streams[g][1]));
        }

        // Step 3
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
            cudaErrorHandle(hipEventSynchronize(events[g][2]));
        }
        
        // Step 4
        for(int g = 0; g < gpus; g++){
            double *mat_change = mat_gpu[g];
            mat_gpu[g] = mat_gpu_tmp[g];
            mat_gpu_tmp[g] = mat_change;
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}


/* 
void no_kernel_overlap(double **mat_gpu, double **mat_gpu_tmp, int width, int height, int depth, int iter, int gpus, int *slices_device, dim3 gridDim, dim3 blockDim, void*** kernelCollEdge, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Transfer 2 slice of the matrix
        for(int g = 1; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (slices_device[g-1]-1)*width + 1, g-1, mat_gpu_tmp[g] + width + 1, g, (width-2)*(height-2)*sizeof(double), streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }
        // Transfers n-2 slice of the matrix
        for(int g = 0; g < gpus-1; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + 1, g+1, mat_gpu_tmp[g] + (slices_device[g]-2)*width + 1, g, (width-2)*(height-2)*sizeof(double), streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
        }

        // Step 3
        for (int g = 0; g < gpus; g++) {
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
        }
        
        // Step 4
        for(int g = 0; g < gpus; g++){
            double *mat_change = mat_gpu[g];
            mat_gpu[g] = mat_gpu_tmp[g];
            mat_gpu_tmp[g] = mat_change;
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));
    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void no_kernel_nooverlap(double **mat_gpu, double **mat_gpu_tmp, int width, int height, int depth, int iter, int gpus, int *slices_device, dim3 gridDim, dim3 blockDim, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 2
        // Transfers 2 slice of the matrix
        for(int g = 1; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (slices_device[g-1]-1)*width + 1, g-1, mat_gpu_tmp[g] + width + 1, g, (width-2)*(height-2)*sizeof(double), streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }
        // Transfers n-2 slice of the matrix
        for(int g = 0; g < gpus-1; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + 1, g+1, mat_gpu_tmp[g] + (slices_device[g]-2)*width + 1, g, (width-2)*(height-2)*sizeof(double), streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
        }

        // Step 3
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
        }
        
        // Step 4
        for(int g = 0; g < gpus; g++){
            double *mat_change = mat_gpu[g];
            mat_gpu[g] = mat_gpu_tmp[g];
            mat_gpu_tmp[g] = mat_change;
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void no_communication_overlap(double **mat_gpu, double **mat_gpu_tmp, int width, int height, int depth, int iter, int gpus, int *slices_device, dim3 gridDim, dim3 blockDim, void*** kernelCollEdge, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 1
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiEdge, gridDim, blockDim, kernelCollEdge[g], 0, streams[g][0]));
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][0]));
        }

        for (int g = 0; g < gpus; g++) {
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
        }
        
        for(int g = 0; g < gpus; g++){
            double *mat_change = mat_gpu[g];
            mat_gpu[g] = mat_gpu_tmp[g];
            mat_gpu_tmp[g] = mat_change;
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));
    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void no_communication_nooverlap(double **mat_gpu, double **mat_gpu_tmp, int width, int height, int depth, int iter, int gpus, int *slices_device, dim3 gridDim, dim3 blockDim, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 1
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }

        // Step 3
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
        }
        
        // Step 4
        for(int g = 0; g < gpus; g++){
            double *mat_change = mat_gpu[g];
            mat_gpu[g] = mat_gpu_tmp[g];
            mat_gpu_tmp[g] = mat_change;
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void only_events(double **mat_gpu, double **mat_gpu_tmp, int width, int height, int depth, int iter, int gpus, int *slices_device, dim3 gridDim, dim3 blockDim, void*** kernelCollMid){

    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
         for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }

        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
        }

        for(int g = 0; g < gpus; g++){
            double *mat_change = mat_gpu[g];
            mat_gpu[g] = mat_gpu_tmp[g];
            mat_gpu_tmp[g] = mat_change;
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void only_calculation_overlap(double **mat_gpu, double **mat_gpu_tmp, int width, int height, int depth, int iter, int gpus, int *slices_device, dim3 gridDim, dim3 blockDim, void*** kernelCollEdge, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 1
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            // Computes the upper and lower slice
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiEdge, gridDim, blockDim, kernelCollEdge[g], 0, streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
            // Computes the rest of the slices
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
        }

        // Step 3
        for (int g = 0; g < gpus; g++) {
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
        }
        
        iter--;
    }
    

    cudaErrorHandle(hipEventRecord(stopevent));
    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void only_calculation_nooverlap(double **mat_gpu, double **mat_gpu_tmp, int width, int height, int depth, int iter, int gpus, int *slices_device, dim3 gridDim, dim3 blockDim, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 1
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[g], 0, streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }

        // Step 3
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
        }

        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void only_communication_overlap(double **mat_gpu, double **mat_gpu_tmp, int width, int height, int depth, int iter, int gpus, int *slices_device, dim3 gridDim, dim3 blockDim, void*** kernelCollEdge, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Transfer 2 slice of the matrix
        for(int g = 1; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (slices_device[g-1]-1)*width + 1, g-1, mat_gpu_tmp[g] + width + 1, g, (width-2)*(height-2)*sizeof(double), streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }
        // Transfers n-2 slice of the matrix
        for(int g = 0; g < gpus-1; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + 1, g+1, mat_gpu_tmp[g] + (slices_device[g]-2)*width + 1, g, (width-2)*(height-2)*sizeof(double), streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
        }

        // Step 3
        for (int g = 0; g < gpus; g++) {
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));
    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}



void only_communication_nooverlap(double **mat_gpu, double **mat_gpu_tmp, int width, int height, int depth, int iter, int gpus, int *slices_device, dim3 gridDim, dim3 blockDim, void*** kernelCollMid){
    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    while(iter > 0){
        // Step 2
        // Transfers 2 slice of the matrix
        for(int g = 1; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g-1] + (slices_device[g-1]-1)*width + 1, g-1, mat_gpu_tmp[g] + width + 1, g, (width-2)*(height-2)*sizeof(double), streams[g][0]));
            cudaErrorHandle(hipEventRecord(events[g][0], streams[g][0]));
        }
        // Transfers n-2 slice of the matrix
        for(int g = 0; g < gpus-1; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipMemcpyPeerAsync(mat_gpu_tmp[g+1] + 1, g+1, mat_gpu_tmp[g] + (slices_device[g]-2)*width + 1, g, (width-2)*(height-2)*sizeof(double), streams[g][1]));
            cudaErrorHandle(hipEventRecord(events[g][1], streams[g][1]));
        }

        // Step 3
        for(int g = 0; g < gpus; g++){
            cudaErrorHandle(hipSetDevice(g));
            cudaErrorHandle(hipEventSynchronize(events[g][0]));
            cudaErrorHandle(hipEventSynchronize(events[g][1]));
        }
        iter--;
    }
    cudaErrorHandle(hipEventRecord(stopevent));

    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}

 */