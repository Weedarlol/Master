#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <nvtx3/nvToolsExt.h>

#include "programs/jacobi.h"
#include "../../functions/global_functions.h"
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;
namespace cg = cooperative_groups;






void start(int width, int height, int iter, double eps, double dx, double dy, int compare, dim3 blockDim, dim3 gridDim){
    /*
    Variables   | Type  | Description
    total       | int   | Total number of elements in the matrix
    dataPerGpu  | int   | Number of elements per available gpu

    start       |clock_t| Start timer of program
    end         |clock_t| End timer of program

    mat         |*double | Pointer to the allocated matrix in the CPU
    mat_gpu     |**double| Pointer to an allocated matrix in the GPU
    mat_gpu_tmp |**double| Pointer to an allocated matrix in the GPU

    maxEps      |*int   | Pointer to an allocated vector in the GPU used for checking if the matrix is in an acceptable state
    
    comp_suc    |*int   | Checks if the computation is successfull or not
    */

    int total = width*height;
    int print_iter = iter;
    clock_t start, end;


    double *mat, *mat_gpu, *mat_gpu_tmp;
    cudaErrorHandle(hipHostMalloc(&mat, total*sizeof(double)));
    cudaErrorHandle(hipMalloc(&mat_gpu, total*sizeof(double*)));
    cudaErrorHandle(hipMalloc(&mat_gpu_tmp, total*sizeof(double*)));
    

    int *maxEps, *comp_suc;;
    cudaErrorHandle(hipMalloc(&maxEps, blockDim.x*blockDim.y*blockDim.z*gridDim.x*gridDim.y*gridDim.z*sizeof(int)));
    cudaErrorHandle(hipHostMalloc(&comp_suc, sizeof(int*)));


    /* initialization */
    fillValues(mat, dx, dy, width, height);
    





    // Here we are done with the allocation, and start with the compution
    start = clock();

    // Copies elemts over from CPU to the device.
    cudaErrorHandle(hipMemcpyAsync(mat_gpu, mat, total*sizeof(double), hipMemcpyHostToDevice));
    cudaErrorHandle(hipMemsetAsync(mat_gpu_tmp, 0, total*sizeof(double)));

    // Creates an array where its elements are features in hipLaunchCooperativeKernel
    void *kernelArgs[] = {&mat_gpu, &mat_gpu_tmp, &eps, &width, &height, &iter, &maxEps};


    // Runs device
    cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobi, gridDim, blockDim, kernelArgs));

    cudaErrorHandle(hipDeviceSynchronize());

    // Copies back value from device i to CPU
    cudaErrorHandle(hipMemcpy(mat, mat_gpu, total*sizeof(double), hipMemcpyDeviceToHost));
    
    cudaErrorHandle(hipMemcpy(comp_suc, maxEps, sizeof(int*), hipMemcpyDeviceToHost));

    cudaErrorHandle(hipDeviceSynchronize());

    end = clock();







    if(*comp_suc != 0){
        printf("The computation found a solution. It computed it within %i iterations (%i - %i) and %.3f seconds.\nWidth = %i, Height = %i\nthreadBlock = (%d, %d, %d), gridDim = (%d, %d, %d)\n\n", 
        print_iter - *comp_suc, print_iter, *comp_suc, ((double) (end - start)) / CLOCKS_PER_SEC, width, height, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
    }
    else{
        printf("The computation did not find a solution after all its iterations, it ran = %i iterations (%i - %i). It completed it in %.3f seconds.\nWidth = %i, Height = %i\nthreadBlock = (%d, %d, %d), gridDim = (%d, %d, %d)\n\n", 
        print_iter - *comp_suc, print_iter, *comp_suc, ((double) (end - start)) / CLOCKS_PER_SEC, width, height, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
    }


    // Used to compare the matrix to the matrix which only the CPU created
    if(compare == 1){
        double* mat_compare = (double*)malloc(width * height * sizeof(double));
        FILE *fptr;
        char filename[30];
        sprintf(filename, "../CPU/CPUMatrix%i_%i.txt", width, height);

        printf("Comparing the matrixes\n");

        fptr = fopen(filename, "r");
        if (fptr == NULL) {
            printf("Error opening file.\n");
            exit(EXIT_FAILURE);
        }

        // Read matrix values from the file
        for (int i = 0; i < height; i++) {
            for (int j = 0; j < width; j++) {
                if (fscanf(fptr, "%lf", &mat_compare[j + i * width]) != 1) {
                    printf("Error reading from file.\n");
                    fclose(fptr);
                    free(mat_compare);
                    exit(EXIT_FAILURE);
                }
            }
        }

        fclose(fptr);


        // Comparing the elements
        for (int i = 1; i < height-1; i++) {
            for (int j = 1; j < width-1; j++) {
                if (fabs(mat[j + i * width] - mat_compare[j + i * width]) > 1e-15)  {
                    printf("Mismatch found at position (%d, %d) (%.16f, %.16f)\n", i, j, mat[j + i * width], mat_compare[j + i * width]);
                    free(mat_compare);
                    exit(EXIT_FAILURE);
                    cudaErrorHandle(hipDeviceSynchronize());
                }
            }
        }


        printf("All elements match!\n");
        

        // Free allocated memory
        free(mat_compare);
    }
    

    cudaErrorHandle(hipHostFree(mat));
    cudaErrorHandle(hipFree(mat_gpu));
    cudaErrorHandle(hipFree(mat_gpu_tmp));
}



int main(int argc, char *argv[]) {
    /*
    Functions   | Type           | Input
    start       | void           | int width, int height, int iter, double eps,
                                   double dx, double dy, dim3 blockDim,
                                   dim3 gridDim

    fillValues  | void           | double *mat, double dx, double dy, int width,
                                   int height

    jacobi      |__global__ void | double *mat_gpu, double *mat_tmp, double eps,
                                   int width, int height, int iter

    ____________________________________________________________________________
    Variables   | Type  | Description
    width       | int   | The width of the matrix
    height      | int   | The height of the matrix
    iter        | int   | Number of max iterations for the jacobian algorithm

    eps         | double | The limit for accepting the state of the matrix during jacobian algorithm
    dx          | double | Distance between each element in the matrix in x direction
    dy          | double | Distance between each element in the matrix in y direction

    blockDim    | dim3  | Number of threads in 3 directions for each block
    gridDim     | dim3  | Number of blocks in 3 directions for the whole grid
    */
    if (argc != 5) {
        printf("Usage: %s <Width> <Height> <Iterations>", argv[0]); // Programname
        return 1;
    }

    int width = atoi(argv[1]);
    int height = atoi(argv[2]);
    int iter = atoi(argv[3]);
    int compare = atoi(argv[4]);

    double eps = 1.0e-14;
    double dx = 2.0 / (width - 1);
    double dy = 2.0 / (height - 1);

    dim3 blockDim(32, 32, 1);
    dim3 gridDim(16, 1, 1);

    start(width, height, iter, eps, dx, dy, compare, blockDim, gridDim);

    return 0;
}
