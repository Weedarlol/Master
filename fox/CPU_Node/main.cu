#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>

#include "mpi.h"


void fillValues3D(double *mat, int width, int height, int depth_node, double dx, double dy, double dz) {
    double x, y, z;

    // Assuming the data in the matrix is stored contiguously in memory
    memset(mat, 0, height * width * depth_node * sizeof(double));

    for (int i = 1; i < depth_node-1; i++) {
        z = i * dz; // z coordinate
        for (int j = 1; j < height - 1; j++) {
            y = j * dy; // z coordinate
            for (int k = 1; k < width - 1; k++) {
                x = k * dx; // x coordinate
                mat[k +  j*width + i*width*height] = sin(M_PI * x) * sin(M_PI * y) * sin(M_PI * z);
            }
        }
    }
}

int main(int argc, char *argv[]) {
    /*
    width       | int   | The width of the matrix
    height      | int   | The height of the matrix
    iter        | int   | Number of max iterations for the jacobian algorithm

    eps         | double | The limit for accepting the state of the matrix during jacobian algorithm
    maxdelta    | double | The largest difference in the matrix between an iteration
    dx          | double | Distance between each element in the matrix in x direction
    dy          | double | Distance between each element in the matrix in y direction

    mat         |*double | Pointer to the matrix
    mat_tmp     |*double | Pointer to the matrix
    */

    int rank, size;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    if (argc != 5) {
        printf("Wrong number of inputs\n Required inputs: %s <Width> <Height> <Depth> <Iterations> <Node>", argv[0]); // Programname
        return 1;
    }

    int width = atoi(argv[1]);
    int height = atoi(argv[2]);
    int depth = atoi(argv[3]);
    int iter = atoi(argv[4]);
    int print_iter = iter;
    int depth_node = depth/size + 1;

    double dx = 2.0 / (width - 1);
    double dy = 2.0 / (height - 1);
    double dz = 2.0 / (depth - 1);

    double *mat;
    double *mat_tmp;

    clock_t start, end;

    mat = (double*)malloc(width*height*depth_node*sizeof(double));
    mat_tmp = (double*)malloc(width*height*depth_node*sizeof(double));

    /* initialization */
    fillValues3D(mat, width, height, depth_node, dx, dy, dz);

    start = clock();
    double division = 1/6.0;

    /* if(rank == 0){
        MPI_Send(&mat[width*height*(depth_node-1)], width*height, MPI_DOUBLE, 1, 0, MPI_COMM_WORLD);
    }
    else if(rank == 1){
        MPI_Recv(&mat, width*height, MPI_DOUBLE, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    } */

    /* Performing Jacobian Matrix Calculation */
    // Performing a number of iterations while statement is not satisfied
    /* while (iter > 0) {
        for(int i = 1; i < depth_node - 1; i++){
            for(int j = 1; j < height - 1; j++){
                for(int k = 1; k < width - 1; k++) {
                    int index = k + j * width + i * width * height;
                    mat_tmp[index] = division * (
                    mat[index + 1]            + mat[index - 1] +
                    mat[index + width]        + mat[index - width] +
                    mat[index + width*height] + mat[index - width*height]);
                }
            }
        }
        

        iter--;

        double *mat_tmp_swap = mat_tmp;
        mat_tmp = mat;
        mat = mat_tmp_swap;
    } */

    end = clock();

    if(rank == 0){
        for(int i = 0; i < depth_node; i++){
            for(int j = 0; j < height; j++){
                for(int k = 0; k < width; k++){
                    printf("%.6f ", mat[k + j*width + i*width*height]);
                }
                printf("\n");
            }
            printf("\n\n");
        }
        printf("\n\n\n");
    }


    if(rank == 0){
        MPI_Send(&mat[width*height*(depth_node-2)], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD);
        MPI_Recv(&mat[width*height*(depth_node-1)], width*height, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    }
    else if(rank == 1){
        MPI_Recv(&mat,                              width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        MPI_Send(&mat[width*height],                width*height, MPI_DOUBLE, rank-1, 0, MPI_COMM_WORLD);
    }

    if(rank == 0){
        for(int i = 0; i < depth_node; i++){
            for(int j = 0; j < height; j++){
                for(int k = 0; k < width; k++){
                    printf("%.6f ", mat[k + j*width + i*width*height]);
                }
                printf("\n");
            }
            printf("\n\n");
        }
    }


    /* if(compare == 1){
        if(rank == 0){
            double* mat_compare = (double*)malloc(width * height * sizeof(double));
            FILE *fptr;
            char filename[30];
            sprintf(filename, "../CPU/CPUMatrix%i_%i.txt", width, height);

            printf("Comparing the matrixes\n");

            fptr = fopen(filename, "r");
            if (fptr == NULL) {
                printf("Error opening file.\n");
                exit(EXIT_FAILURE);
            }

            // Read matrix values from the file
            for (int i = 0; i < height; i++) {
                for (int j = 0; j < width; j++) {
                    if (fscanf(fptr, "%lf", &mat_compare[j + i * width]) != 1) {
                        printf("Error reading from file.\n");
                        fclose(fptr);
                        free(mat_compare);
                        exit(EXIT_FAILURE);
                    }
                }
            }

            fclose(fptr);


            // Comparing the elements
            for (int i = 1; i < height-1; i++) {
                for (int j = 1; j < width-1; j++) {
                    if (fabs(mat[j + i * width] - mat_compare[j + i * width]) > 1e-15)  {
                        printf("Mismatch found at position (%d, %d) (%.16f, %.16f)\n", i, j, mat[j + i * width], mat_compare[j + i * width]);
                        free(mat_compare);
                        exit(EXIT_FAILURE);
                        cudaErrorHandle(hipDeviceSynchronize());
                    }
                }
            }


            printf("All elements match!\n");
            

            // Free allocated memory
            free(mat_compare);
        }
    } */


    free(mat);
    free(mat_tmp);

    
    /* printf("It computed through the whole %i iteration(%i - %i) in %.3f seconds \nWidth = %i, Height = %i, Depth = %i\n", 
    print_iter - iter, print_iter, iter, ((double) (end - start)) / CLOCKS_PER_SEC, width, height, depth); */

    MPI_Finalize();

    return 0;
}
