#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>


#include "errorHandle.h"
#include "jacobi.h"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

void initializeStreamsAndEvents(int gpus, hipStream_t streams[][2], hipEvent_t events[][4], hipEvent_t *startevent, hipEvent_t *stopevent){
    for (int g = 0; g < gpus; g++) {
        hipSetDevice(g);
        cudaErrorHandle(hipStreamCreate(&streams[g][0]));
        cudaErrorHandle(hipStreamCreate(&streams[g][1]));
        cudaErrorHandle(hipEventCreate(&events[g][0]));
        cudaErrorHandle(hipEventCreate(&events[g][1]));
        cudaErrorHandle(hipEventCreate(&events[g][2]));
        cudaErrorHandle(hipEventCreate(&events[g][3]));
    }
    cudaErrorHandle(hipEventCreate(startevent));
    cudaErrorHandle(hipEventCreate(stopevent));
}

void freeStreamsAndEvents(int gpus, hipStream_t streams[][2], hipEvent_t events[][4], hipEvent_t *startevent, hipEvent_t *stopevent) {
    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipStreamDestroy(streams[g][0]));
        cudaErrorHandle(hipStreamDestroy(streams[g][1]));
        cudaErrorHandle(hipEventDestroy(events[g][0]));
        cudaErrorHandle(hipEventDestroy(events[g][1]));
        cudaErrorHandle(hipEventDestroy(events[g][2]));
        cudaErrorHandle(hipEventDestroy(events[g][3]));
    }
}


__global__ void jacobiEdge(hipPitchedPtr mat_gpu, int width, int height, int depth){
    cg::grid_group grid_g = cg::this_grid();
    int thread = grid_g.thread_rank();
    int thread_size = grid_g.size();

    char* d_ptr = static_cast<char*>(mat_gpu.ptr);
    size_t pitch = mat_gpu.pitch;

    int * element  = (int *)(thread);
    element[thread] = thread;
    
}


void full_calculation_overlap(hipPitchedPtr mat_gpu, int width, int height, int depth, int gpus, int iter, dim3 blockDim, dim3 gridDim){
    
    void ***kernelCollMid;
    cudaErrorHandle(hipHostMalloc(&kernelCollMid, gpus * sizeof(void**)));
    // Allocates the elements in the kernelCollMid, used for hipLaunchCooperativeKernel as functon variables.
    for (int g = 0; g < gpus; g++) {
        void **kernelArgs = new void*[4];
        kernelArgs[0] = &mat_gpu;     
        kernelArgs[1] = &width;    
        kernelArgs[2] = &height;
        kernelArgs[3] = &depth;

        kernelCollMid[g] = kernelArgs;
    }

    hipStream_t streams[gpus][2];
    hipEvent_t events[gpus][4], startevent, stopevent;
    initializeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    cudaErrorHandle(hipEventRecord(startevent));

    cudaErrorHandle(hipLaunchCooperativeKernel((void*)jacobiMid, gridDim, blockDim, kernelCollMid[0], 0, streams[0][1]));

    cudaErrorHandle(hipEventRecord(stopevent));
    cudaErrorHandle(hipEventSynchronize(stopevent));

    for(int g = 0; g < gpus; g++){
        cudaErrorHandle(hipSetDevice(g));
        cudaErrorHandle(hipDeviceSynchronize());
    }
    
    float milliseconds = 0.0f;
    cudaErrorHandle(hipEventElapsedTime(&milliseconds, startevent, stopevent));
    printf("Time(event) - %.4f, SolutionFound - %s, IterationsComputed - %i\n",
            milliseconds, (iter == 0) ? "No" : "Yes", iter - iter);

    freeStreamsAndEvents(gpus, streams, events, &startevent, &stopevent);
}

